#include "pytorch_cuda_helper.hpp"

#include "modulated_deform_conv_cuda_kernel.cuh"

void modulated_deformable_im2col_cuda(
    const Tensor data_im, const Tensor data_offset, const Tensor data_mask,
    const int batch_size, const int channels, const int height_im,
    const int width_im, const int height_col, const int width_col,
    const int kernel_h, const int kenerl_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int deformable_group, Tensor data_col) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels = channels * batch_size * height_col * width_col;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "modulated_deformable_im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t *data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t *data_col_ = data_col.data_ptr<scalar_t>();

        modulated_deformable_im2col_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_im_, data_offset_, data_mask_, height_im,
            width_im, kernel_h, kenerl_w, pad_h, pad_w, stride_h, stride_w,
            dilation_h, dilation_w, channel_per_deformable_group, batch_size,
            channels, deformable_group, height_col, width_col, data_col_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void modulated_deformable_col2im_cuda(
    const Tensor data_col, const Tensor data_offset, const Tensor data_mask,
    const int batch_size, const int channels, const int height_im,
    const int width_im, const int height_col, const int width_col,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int deformable_group, Tensor grad_im) {
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels =
      channels * kernel_h * kernel_w * batch_size * height_col * width_col;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "modulated_deformable_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t *data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        modulated_deformable_col2im_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, data_offset_, data_mask_, channels,
            height_im, width_im, kernel_h, kernel_w, pad_h, pad_w, stride_h,
            stride_w, dilation_h, dilation_w, channel_per_deformable_group,
            batch_size, deformable_group, height_col, width_col, grad_im_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void modulated_deformable_col2im_coord_cuda(
    const Tensor data_col, const Tensor data_im, const Tensor data_offset,
    const Tensor data_mask, const int batch_size, const int channels,
    const int height_im, const int width_im, const int height_col,
    const int width_col, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int deformable_group,
    Tensor grad_offset, Tensor grad_mask) {
  const int num_kernels = batch_size * height_col * width_col * 2 * kernel_h *
                          kernel_w * deformable_group;
  const int channel_per_deformable_group =
      channels * kernel_h * kernel_w / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "modulated_deformable_col2im_coord_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t *data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t *grad_offset_ = grad_offset.data_ptr<scalar_t>();
        scalar_t *grad_mask_ = grad_mask.data_ptr<scalar_t>();

        modulated_deformable_col2im_coord_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, data_im_, data_offset_, data_mask_,
            channels, height_im, width_im, kernel_h, kernel_w, pad_h, pad_w,
            stride_h, stride_w, dilation_h, dilation_w,
            channel_per_deformable_group, batch_size,
            2 * kernel_h * kernel_w * deformable_group, deformable_group,
            height_col, width_col, grad_offset_, grad_mask_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void ModulatedDeformConvForwardCUDAKernelLauncher(
    Tensor input, Tensor weight, Tensor bias, Tensor ones, Tensor offset,
    Tensor mask, Tensor output, Tensor columns, int kernel_h, int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, const int group,
    const int deformable_group, const bool with_bias) {
  at::DeviceGuard guard(input.device());

  const int batch = input.size(0);
  const int channels = input.size(1);
  const int height = input.size(2);
  const int width = input.size(3);

  const int channels_out = weight.size(0);
  const int channels_kernel = weight.size(1);
  const int kernel_h_ = weight.size(2);
  const int kernel_w_ = weight.size(3);

  if (kernel_h_ != kernel_h || kernel_w_ != kernel_w)
    AT_ERROR("Input shape and kernel shape wont match: (%d x %d vs %d x %d).",
             kernel_h_, kernel_w, kernel_h_, kernel_w_);
  if (channels != channels_kernel * group)
    AT_ERROR("Input shape and kernel channels wont match: (%d vs %d).",
             channels, channels_kernel * group);

  const int height_out =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

  if (ones.ndimension() != 2 ||
      ones.size(0) * ones.size(1) < height_out * width_out) {
    // Resize plane and fill with ones...
    ones = at::ones({height_out, width_out}, input.options());
  }

  // resize output
  output = output.view({batch, channels_out, height_out, width_out}).zero_();
  // resize temporary columns
  columns =
      at::zeros({channels * kernel_h * kernel_w, 1 * height_out * width_out},
                input.options());

  output = output.view({output.size(0), group, output.size(1) / group,
                        output.size(2), output.size(3)});

  for (int b = 0; b < batch; b++) {
    modulated_deformable_im2col_cuda(
        input[b], offset[b], mask[b], 1, channels, height, width, height_out,
        width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w,
        dilation_h, dilation_w, deformable_group, columns);

    // divide into group
    weight = weight.view({group, weight.size(0) / group, weight.size(1),
                          weight.size(2), weight.size(3)});
    columns = columns.view({group, columns.size(0) / group, columns.size(1)});

    for (int g = 0; g < group; g++) {
      output[b][g] = output[b][g]
                         .flatten(1)
                         .addmm_(weight[g].flatten(1), columns[g])
                         .view_as(output[b][g]);
    }

    weight = weight.view({weight.size(0) * weight.size(1), weight.size(2),
                          weight.size(3), weight.size(4)});
    columns =
        columns.view({columns.size(0) * columns.size(1), columns.size(2)});
  }

  output = output.view({output.size(0), output.size(1) * output.size(2),
                        output.size(3), output.size(4)});

  if (with_bias) {
    output += bias.view({1, bias.size(0), 1, 1});
  }
}

void ModulatedDeformConvBackwardCUDAKernelLauncher(
    Tensor input, Tensor weight, Tensor bias, Tensor ones, Tensor offset,
    Tensor mask, Tensor columns, Tensor grad_input, Tensor grad_weight,
    Tensor grad_bias, Tensor grad_offset, Tensor grad_mask, Tensor grad_output,
    int kernel_h, int kernel_w, int stride_h, int stride_w, int pad_h,
    int pad_w, int dilation_h, int dilation_w, int group, int deformable_group,
    const bool with_bias) {
  at::DeviceGuard guard(input.device());

  const int batch = input.size(0);
  const int channels = input.size(1);
  const int height = input.size(2);
  const int width = input.size(3);

  const int channels_kernel = weight.size(1);
  const int kernel_h_ = weight.size(2);
  const int kernel_w_ = weight.size(3);
  if (kernel_h_ != kernel_h || kernel_w_ != kernel_w)
    AT_ERROR("Input shape and kernel shape wont match: (%d x %d vs %d x %d).",
             kernel_h_, kernel_w, kernel_h_, kernel_w_);
  if (channels != channels_kernel * group)
    AT_ERROR("Input shape and kernel channels wont match: (%d vs %d).",
             channels, channels_kernel * group);

  const int height_out =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  const int width_out =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

  if (ones.ndimension() != 2 ||
      ones.size(0) * ones.size(1) < height_out * width_out) {
    // Resize plane and fill with ones...
    ones = at::ones({height_out, width_out}, input.options());
  }

  grad_input = grad_input.view({batch, channels, height, width});
  columns = at::zeros({channels * kernel_h * kernel_w, height_out * width_out},
                      input.options());

  grad_output =
      grad_output.view({grad_output.size(0), group, grad_output.size(1) / group,
                        grad_output.size(2), grad_output.size(3)});

  for (int b = 0; b < batch; b++) {
    // divide int group
    columns = columns.view({group, columns.size(0) / group, columns.size(1)});
    weight = weight.view({group, weight.size(0) / group, weight.size(1),
                          weight.size(2), weight.size(3)});

    for (int g = 0; g < group; g++) {
      columns[g].addmm_(weight[g].flatten(1).transpose(0, 1),
                        grad_output[b][g].flatten(1), 0.0f, 1.0f);
    }

    columns =
        columns.view({columns.size(0) * columns.size(1), columns.size(2)});
    weight = weight.view({weight.size(0) * weight.size(1), weight.size(2),
                          weight.size(3), weight.size(4)});

    // gradient w.r.t. input coordinate data
    modulated_deformable_col2im_coord_cuda(
        columns, input[b], offset[b], mask[b], 1, channels, height, width,
        height_out, width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h,
        stride_w, dilation_h, dilation_w, deformable_group, grad_offset[b],
        grad_mask[b]);
    // gradient w.r.t. input data
    modulated_deformable_col2im_cuda(
        columns, offset[b], mask[b], 1, channels, height, width, height_out,
        width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w,
        dilation_h, dilation_w, deformable_group, grad_input[b]);

    // gradient w.r.t. weight, dWeight should accumulate across the batch and
    // group
    modulated_deformable_im2col_cuda(
        input[b], offset[b], mask[b], 1, channels, height, width, height_out,
        width_out, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w,
        dilation_h, dilation_w, deformable_group, columns);

    columns = columns.view({group, columns.size(0) / group, columns.size(1)});
    grad_weight = grad_weight.view({group, grad_weight.size(0) / group,
                                    grad_weight.size(1), grad_weight.size(2),
                                    grad_weight.size(3)});
    if (with_bias)
      grad_bias = grad_bias.view({group, grad_bias.size(0) / group});

    for (int g = 0; g < group; g++) {
      grad_weight[g] =
          grad_weight[g]
              .flatten(1)
              .addmm_(grad_output[b][g].flatten(1), columns[g].transpose(0, 1))
              .view_as(grad_weight[g]);
      if (with_bias) {
        grad_bias[g] =
            grad_bias[g]
                .view({-1, 1})
                .addmm_(grad_output[b][g].flatten(1), ones.view({-1, 1}))
                .view(-1);
      }
    }

    columns =
        columns.view({columns.size(0) * columns.size(1), columns.size(2)});
    grad_weight = grad_weight.view({grad_weight.size(0) * grad_weight.size(1),
                                    grad_weight.size(2), grad_weight.size(3),
                                    grad_weight.size(4)});
    if (with_bias)
      grad_bias = grad_bias.view({grad_bias.size(0) * grad_bias.size(1)});
  }
  grad_output = grad_output.view({grad_output.size(0) * grad_output.size(1),
                                  grad_output.size(2), grad_output.size(3),
                                  grad_output.size(4)});
}
