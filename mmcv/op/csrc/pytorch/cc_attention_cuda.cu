// Modified from
// https://github.com/LikeLy-Journey/SegmenTron/blob/master/segmentron/modules/csrc/criss_cross_attention/ca_cuda.cu

#include <THC/THC.h>

#include <THC/THCDeviceUtils.cuh>

#include "pytorch_cuda_helper.hpp"

#include "cc_attention_cuda_kernel.cuh"

void CAForwardCUDAKernelLauncher(const Tensor t, const Tensor f,
                                 Tensor weight) {
  AT_ASSERTM(t.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(f.device().is_cuda(), "input must be a CUDA tensor");

  auto n = t.size(0);
  auto c = t.size(1);
  auto h = t.size(2);
  auto w = t.size(3);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Run kernel
  dim3 threads(32, 32);
  int d1 = (w + threads.x - 1) / threads.x;
  int d2 = (h + threads.y - 1) / threads.y;
  int d3 = h + w;
  dim3 blocks(d1, d2, d3);

  AT_DISPATCH_FLOATING_TYPES(t.scalar_type(), "ca_forward", [&] {
    ca_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        t.contiguous().data_ptr<scalar_t>(),
        f.contiguous().data_ptr<scalar_t>(),
        weight.contiguous().data_ptr<scalar_t>(), n, c, h, w);
  });
  THCudaCheck(hipGetLastError());
}

void CABackwardCUDAKernelLauncher(const Tensor dw, const Tensor t,
                                  const Tensor f, Tensor dt, Tensor df) {
  AT_ASSERTM(dw.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(t.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(f.device().is_cuda(), "input must be a CUDA tensor");

  auto n = t.size(0);
  auto c = t.size(1);
  auto h = t.size(2);
  auto w = t.size(3);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Run kernel
  dim3 threads(32, 32);
  int d1 = (w + threads.x - 1) / threads.x;
  int d2 = (h + threads.y - 1) / threads.y;
  int d3 = c;
  dim3 blocks(d1, d2, d3);

  AT_DISPATCH_FLOATING_TYPES(t.scalar_type(), "ca_backward_kernel_t", [&] {
    ca_backward_kernel_t<scalar_t><<<blocks, threads, 0, stream>>>(
        dw.contiguous().data_ptr<scalar_t>(),
        t.contiguous().data_ptr<scalar_t>(),
        f.contiguous().data_ptr<scalar_t>(),
        dt.contiguous().data_ptr<scalar_t>(), n, c, h, w);
  });

  AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "ca_backward_kernel_f", [&] {
    ca_backward_kernel_f<scalar_t><<<blocks, threads, 0, stream>>>(
        dw.contiguous().data_ptr<scalar_t>(),
        t.contiguous().data_ptr<scalar_t>(),
        f.contiguous().data_ptr<scalar_t>(),
        df.contiguous().data_ptr<scalar_t>(), n, c, h, w);
  });
  THCudaCheck(hipGetLastError());
}

void CAMapForwardCUDAKernelLauncher(const Tensor weight, const Tensor g,
                                    Tensor out) {
  AT_ASSERTM(weight.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(g.device().is_cuda(), "input must be a CUDA tensor");

  auto n = g.size(0);
  auto c = g.size(1);
  auto h = g.size(2);
  auto w = g.size(3);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Run kernel
  dim3 threads(32, 32);
  int d1 = (w + threads.x - 1) / threads.x;
  int d2 = (h + threads.y - 1) / threads.y;
  int d3 = c;
  dim3 blocks(d1, d2, d3);

  AT_DISPATCH_FLOATING_TYPES(g.scalar_type(), "ca_map_forward", [&] {
    ca_map_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        weight.contiguous().data_ptr<scalar_t>(),
        g.contiguous().data_ptr<scalar_t>(),
        out.contiguous().data_ptr<scalar_t>(), n, c, h, w);
  });
  THCudaCheck(hipGetLastError());
}

void CAMapBackwardCUDAKernelLauncher(const Tensor dout, const Tensor weight,
                                     const Tensor g, Tensor dw, Tensor dg) {
  AT_ASSERTM(dout.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(weight.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(g.device().is_cuda(), "input must be a CUDA tensor");

  auto n = dout.size(0);
  auto c = dout.size(1);
  auto h = dout.size(2);
  auto w = dout.size(3);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Run kernel
  dim3 threads(32, 32);
  int d1 = (w + threads.x - 1) / threads.x;
  int d2 = (h + threads.y - 1) / threads.y;
  int d3 = h + w;
  dim3 blocks(d1, d2, d3);

  AT_DISPATCH_FLOATING_TYPES(
      weight.scalar_type(), "ca_map_backward_kernel_w", [&] {
        ca_map_backward_kernel_w<scalar_t><<<blocks, threads, 0, stream>>>(
            dout.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            g.contiguous().data_ptr<scalar_t>(),
            dw.contiguous().data_ptr<scalar_t>(), n, c, h, w);
      });

  AT_DISPATCH_FLOATING_TYPES(g.scalar_type(), "ca_map_backward_kernel_g", [&] {
    ca_map_backward_kernel_g<scalar_t><<<blocks, threads, 0, stream>>>(
        dout.contiguous().data_ptr<scalar_t>(),
        weight.contiguous().data_ptr<scalar_t>(),
        g.contiguous().data_ptr<scalar_t>(),
        dg.contiguous().data_ptr<scalar_t>(), n, c, h, w);
  });
  THCudaCheck(hipGetLastError());
}
