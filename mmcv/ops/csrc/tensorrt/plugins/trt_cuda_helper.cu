#include "hip/hip_runtime.h"
#include "common_cuda_helper.hpp"
#include "trt_cuda_helper.cuh"
#include "trt_plugin_helper.hpp"

using mmcv::TensorDesc;

template <class scalar_t>
__global__ void copy_permute_kernel(scalar_t *dst, const scalar_t *src, int n,
                                    TensorDesc ts_src_stride,
                                    TensorDesc ts_dst_stride,
                                    TensorDesc ts_permute) {
  const int src_dim = ts_src_stride.dim;
  int *src_stride = &(ts_src_stride.stride[0]);
  int *dst_stride = &(ts_dst_stride.stride[0]);
  int *permute = &(ts_permute.shape[0]);
  CUDA_1D_KERNEL_LOOP(index, n) {
    size_t dst_index = index;
    size_t src_index = 0;
    for (int i = 0; i < src_dim; ++i) {
      int dim_index = dst_index / dst_stride[i];
      dst_index = dst_index % dst_stride[i];
      src_index += dim_index * src_stride[permute[i]];
    }
    dst[index] = src[src_index];
  }
}

template <class scalar_t>
void memcpyPermute(scalar_t *dst, const scalar_t *src, int *src_size,
                   int *permute, int src_dim, hipStream_t stream) {
  size_t copy_size = 1;
  TensorDesc ts_permute;
  memcpy(&(ts_permute.shape[0]), permute, src_dim * sizeof(int));

  TensorDesc ts_src_stride;
  TensorDesc ts_dst_stride;
  ts_src_stride.dim = src_dim;
  ts_dst_stride.dim = src_dim;
  int *src_stride = &(ts_src_stride.stride[0]);
  int *dst_stride = &(ts_dst_stride.stride[0]);
  int *dst_size = &(ts_dst_stride.shape[0]);
  src_stride[src_dim - 1] = 1;
  dst_stride[src_dim - 1] = 1;

  for (int i = src_dim - 1; i >= 0; --i) {
    dst_size[i] = src_size[permute[i]];
    if (i < src_dim - 1) {
      src_stride[i] = src_stride[i + 1] * src_size[i + 1];
    }
  }

  for (int i = src_dim - 1; i >= 0; --i) {
    copy_size *= dst_size[i];
    if (i < src_dim - 1) {
      dst_stride[i] = dst_stride[i + 1] * dst_size[i + 1];
    }
  }

  copy_permute_kernel<scalar_t>
      <<<GET_BLOCKS(copy_size), THREADS_PER_BLOCK, 0, stream>>>(
          dst, src, copy_size, ts_src_stride, ts_dst_stride, ts_permute);
}

template void memcpyPermute<float>(float *dst, const float *src, int *src_size,
                                   int *permute, int src_dim,
                                   hipStream_t stream);
