#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved
#include "common_cuda_helper.hpp"
#include "trt_cuda_helper.cuh"
#include "trt_plugin_helper.hpp"

template <typename scalar_t>
__global__ void top_bottom_pool_kernel(const scalar_t *input, scalar_t *output,
                                       const int batch_size, const int channels,
                                       const int height, const int width,
                                       const int pool_type) {
  const int nthreads = batch_size * channels * width;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n_idx = index / (channels * width);  // batch
    int w_idx = index % width;               // width
    int c_idx = (index / width) % channels;  // channels
    int offset_n = n_idx * channels * width * height;
    int offset_n_c = offset_n + c_idx * width * height;
    int direction = -1;            // in [-1, 1], default for TopPool
    int index_start = height - 2;  // default for TopPool
    // pool_type in [0, 1]
    if (pool_type == 0) {
      // TopPool
      // directly copy the most bottom value from input to output
      output[offset_n_c + (height - 1) * width + w_idx] =
          input[offset_n_c + (height - 1) * width + w_idx];
    } else {
      // BottomPool
      // directly copy the most top value from input to output
      output[offset_n_c + w_idx] = input[offset_n_c + w_idx];
      index_start = 1;
      direction = 1;
    }
    // do pool
    for (int h = index_start; h >= 0 && h < height; h += direction) {
      output[offset_n_c + h * width + w_idx] =
          max(output[offset_n_c + (h - direction) * width + w_idx],
              input[offset_n_c + h * width + w_idx]);
    }
  }
}

template <typename scalar_t>
__global__ void left_right_pool_kernel(const scalar_t *input, scalar_t *output,
                                       const int batch_size, const int channels,
                                       const int height, const int width,
                                       const int pool_type) {
  const int nthreads = batch_size * channels * height;
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n_idx = index / (channels * height);  // batch
    int h_idx = index % height;               // height
    int c_idx = (index / height) % channels;  // channels
    int offset_n = n_idx * channels * width * height;
    int offset_n_c = offset_n + c_idx * width * height;
    int offset_n_c_h = offset_n_c + h_idx * width;
    int direction = -1;           // in [-1, 1], default for LeftPool
    int index_start = width - 2;  // default for LeftPool
    // pool_type in [2, 3]
    if (pool_type == 2) {
      // LeftPool
      // directly copy the most right value from input to output
      output[offset_n_c_h + width - 1] = input[offset_n_c_h + width - 1];
    } else {
      // RightPool
      // directly copy the most left value from input to output
      output[offset_n_c_h] = input[offset_n_c_h];
      index_start = 1;
      direction = 1;
    }
    // do pool
    for (int w = index_start; w >= 0 && w < width; w += direction) {
      output[offset_n_c_h + w] =
          max(output[offset_n_c_h + w - direction], input[offset_n_c_h + w]);
    }
  }
}

template <typename scalar_t>
void CornerPoolForwardLauncher(const scalar_t *input, scalar_t *output,
                               const int batch_size, const int channels,
                               const int height, const int width,
                               const int pool_type, hipStream_t stream) {
  int nthreads = -1, col_block = -1;

  switch (pool_type) {
    case 0:
    case 1:
      nthreads = batch_size * channels * width;
      col_block = GET_BLOCKS(nthreads, THREADS_PER_BLOCK);
      top_bottom_pool_kernel<scalar_t>
          <<<col_block, THREADS_PER_BLOCK, 0, stream>>>(
              input, output, batch_size, channels, height, width, pool_type);
      break;
    case 2:
    case 3:
      nthreads = batch_size * channels * height;
      col_block = GET_BLOCKS(nthreads, THREADS_PER_BLOCK);
      left_right_pool_kernel<scalar_t>
          <<<col_block, THREADS_PER_BLOCK, 0, stream>>>(
              input, output, batch_size, channels, height, width, pool_type);
      break;
  }
}

void CornerPoolForwardLauncher_float(const float *input, float *output,
                                     const int batch_size, const int channels,
                                     const int height, const int width,
                                     const int pool_type, hipStream_t stream) {
  CornerPoolForwardLauncher<float>(input, output, batch_size, channels, height,
                                   width, pool_type, stream);
}
