#include "nms_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

Tensor NMSCUDAKernelLauncher(Tensor boxes, Tensor scores, float iou_threshold,
                             int offset) {
  at::cuda::CUDAGuard device_guard(boxes.device());

  if (boxes.numel() == 0) {
    return at::empty({0}, boxes.options().dtype(at::kLong));
  }
  auto order_t = std::get<1>(scores.sort(0, /*descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);
  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);
  Tensor mask =
      at::empty({boxes_num, col_blocks}, boxes.options().dtype(at::kLong));
  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  nms_cuda<<<blocks, threads, 0, stream>>>(
      boxes_num, iou_threshold, offset, boxes_sorted.data_ptr<float>(),
      (unsigned long long*)mask.data_ptr<int64_t>());

  at::Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host =
      (unsigned long long*)mask_cpu.data_ptr<int64_t>();

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep_t =
      at::zeros({boxes_num}, boxes.options().dtype(at::kBool).device(at::kCPU));
  bool* keep = keep_t.data_ptr<bool>();

  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep[i] = true;
      // set every overlap box with bit 1 in remv
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  AT_CUDA_CHECK(hipGetLastError());
  return order_t.masked_select(keep_t.to(at::kCUDA));
}
