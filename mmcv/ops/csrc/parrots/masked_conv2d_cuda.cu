// Copyright (c) OpenMMLab. All rights reserved
#include "masked_conv2d_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void MaskedIm2colForwardCUDAKernelLauncher(const Tensor bottom_data,
                                           const Tensor mask_h_idx,
                                           const Tensor mask_w_idx,
                                           Tensor top_data, const int kernel_h,
                                           const int kernel_w, const int pad_h,
                                           const int pad_w) {
  int channels = bottom_data.size(1);
  int height = bottom_data.size(2);
  int width = bottom_data.size(3);
  int mask_cnt = mask_h_idx.size(0);
  int output_size = mask_cnt * channels;

  at::cuda::CUDAGuard device_guard(bottom_data.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bottom_data.scalar_type(), "MaskedIm2colLaucherForward", ([&] {
        const scalar_t *bottom_data_ = bottom_data.data_ptr<scalar_t>();
        const int64_t *mask_h_idx_ = mask_h_idx.data_ptr<int64_t>();
        const int64_t *mask_w_idx_ = mask_w_idx.data_ptr<int64_t>();
        scalar_t *top_data_ = top_data.data_ptr<scalar_t>();
        MaskedIm2colForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, bottom_data_, height, width, kernel_h, kernel_w,
                pad_h, pad_w, mask_h_idx_, mask_w_idx_, mask_cnt, top_data_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void MaskedCol2imForwardCUDAKernelLauncher(
    const Tensor bottom_data, const Tensor mask_h_idx, const Tensor mask_w_idx,
    Tensor top_data, const int height, const int width, const int channels) {
  int mask_cnt = mask_h_idx.size(0);
  int output_size = mask_cnt * channels;

  at::cuda::CUDAGuard device_guard(bottom_data.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bottom_data.scalar_type(), "MaskedCol2imLaucherForward", ([&] {
        const scalar_t *bottom_data_ = bottom_data.data_ptr<scalar_t>();
        const int64_t *mask_h_idx_ = mask_h_idx.data_ptr<int64_t>();
        const int64_t *mask_w_idx_ = mask_w_idx.data_ptr<int64_t>();
        scalar_t *top_data_ = top_data.data_ptr<scalar_t>();

        MaskedCol2imForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, bottom_data_, height, width, channels, mask_h_idx_,
                mask_w_idx_, mask_cnt, top_data_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
