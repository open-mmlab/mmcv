#include "carafe_naive_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void CARAFENAIVEForwardCUDAKernelLauncher(const Tensor features,
                                          const Tensor masks, Tensor output,
                                          const int kernel_size,
                                          const int group_size,
                                          const int scale_factor) {
  int output_size = output.numel();
  int channels = output.size(1);
  int height = output.size(2);
  int width = output.size(3);

  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "CARAFENAIVEForward", ([&] {
        carafe_naive_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, features.data_ptr<scalar_t>(),
                masks.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                kernel_size, group_size, scale_factor, channels, height, width);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void CARAFENAIVEBackwardCUDAKernelLauncher(
    const Tensor top_grad, const Tensor features, const Tensor masks,
    Tensor bottom_grad, Tensor mask_grad, const int kernel_size,
    const int group_size, const int scale_factor) {
  int output_size = top_grad.numel();
  int channels = top_grad.size(1);
  int height = top_grad.size(2);
  int width = top_grad.size(3);

  at::cuda::CUDAGuard device_guard(top_grad.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "CARAFENAIVEBackward", ([&] {
        carafe_naive_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, top_grad.data_ptr<scalar_t>(),
                features.data_ptr<scalar_t>(), masks.data_ptr<scalar_t>(),
                bottom_grad.data_ptr<scalar_t>(),
                mask_grad.data_ptr<scalar_t>(), kernel_size, group_size,
                scale_factor, channels, height, width);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}
