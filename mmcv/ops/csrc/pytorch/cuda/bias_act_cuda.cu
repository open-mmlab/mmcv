#include "hip/hip_runtime.h"
// Modified from
// https://github.com/NVlabs/stylegan3/blob/main/torch_utils/ops/bias_act.cpp

// Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <c10/util/Half.h>
#include <hip/hip_runtime.h>
#include <torch/types.h>

#include "pytorch_cuda_helper.hpp"

struct bias_act_kernel_params {
  const void *x;     // [sizeX]
  const void *b;     // [sizeB] or NULL
  const void *xref;  // [sizeX] or NULL
  const void *yref;  // [sizeX] or NULL
  const void *dy;    // [sizeX] or NULL
  void *y;           // [sizeX]

  int grad;
  int act;
  float alpha;
  float gain;
  float clamp;

  int sizeX;
  int sizeB;
  int stepB;
  int loopX;
};

// CUDA kernel selection.

template <class T>
void *choose_bias_act_kernel(const bias_act_kernel_params &p);
//------------------------------------------------------------------------
// Helpers.

template <class T>
struct InternalType;
template <>
struct InternalType<double> {
  typedef double scalar_t;
};
template <>
struct InternalType<float> {
  typedef float scalar_t;
};
template <>
struct InternalType<c10::Half> {
  typedef float scalar_t;
};

//------------------------------------------------------------------------
// CUDA kernel.

template <class T, int A>
__global__ void bias_act_kernel(bias_act_kernel_params p) {
  typedef typename InternalType<T>::scalar_t scalar_t;
  int G = p.grad;
  scalar_t alpha = (scalar_t)p.alpha;
  scalar_t gain = (scalar_t)p.gain;
  scalar_t clamp = (scalar_t)p.clamp;
  scalar_t one = (scalar_t)1;
  scalar_t two = (scalar_t)2;
  scalar_t expRange = (scalar_t)80;
  scalar_t halfExpRange = (scalar_t)40;
  scalar_t seluScale = (scalar_t)1.0507009873554804934193349852946;
  scalar_t seluAlpha = (scalar_t)1.6732632423543772848170429916717;

  // Loop over elements.
  int xi = blockIdx.x * p.loopX * blockDim.x + threadIdx.x;
  for (int loopIdx = 0; loopIdx < p.loopX && xi < p.sizeX;
       loopIdx++, xi += blockDim.x) {
    // Load.
    scalar_t x = (scalar_t)((const T *)p.x)[xi];
    scalar_t b =
        (p.b) ? (scalar_t)((const T *)p.b)[(xi / p.stepB) % p.sizeB] : 0;
    scalar_t xref = (p.xref) ? (scalar_t)((const T *)p.xref)[xi] : 0;
    scalar_t yref = (p.yref) ? (scalar_t)((const T *)p.yref)[xi] : 0;
    scalar_t dy = (p.dy) ? (scalar_t)((const T *)p.dy)[xi] : one;
    scalar_t yy = (gain != 0) ? yref / gain : 0;
    scalar_t y = 0;

    // Apply bias.
    ((G == 0) ? x : xref) += b;

    // linear
    if (A == 1) {
      if (G == 0) y = x;
      if (G == 1) y = x;
    }

    // relu
    if (A == 2) {
      if (G == 0) y = (x > 0) ? x : 0;
      if (G == 1) y = (yy > 0) ? x : 0;
    }

    // lrelu
    if (A == 3) {
      if (G == 0) y = (x > 0) ? x : x * alpha;
      if (G == 1) y = (yy > 0) ? x : x * alpha;
    }

    // tanh
    if (A == 4) {
      if (G == 0) {
        scalar_t c = exp(x);
        scalar_t d = one / c;
        y = (x < -expRange) ? -one : (x > expRange) ? one : (c - d) / (c + d);
      }
      if (G == 1) y = x * (one - yy * yy);
      if (G == 2) y = x * (one - yy * yy) * (-two * yy);
    }

    // sigmoid
    if (A == 5) {
      if (G == 0) y = (x < -expRange) ? 0 : one / (exp(-x) + one);
      if (G == 1) y = x * yy * (one - yy);
      if (G == 2) y = x * yy * (one - yy) * (one - two * yy);
    }

    // elu
    if (A == 6) {
      if (G == 0) y = (x >= 0) ? x : exp(x) - one;
      if (G == 1) y = (yy >= 0) ? x : x * (yy + one);
      if (G == 2) y = (yy >= 0) ? 0 : x * (yy + one);
    }

    // selu
    if (A == 7) {
      if (G == 0)
        y = (x >= 0) ? seluScale * x : (seluScale * seluAlpha) * (exp(x) - one);
      if (G == 1)
        y = (yy >= 0) ? x * seluScale : x * (yy + seluScale * seluAlpha);
      if (G == 2) y = (yy >= 0) ? 0 : x * (yy + seluScale * seluAlpha);
    }

    // softplus
    if (A == 8) {
      if (G == 0) y = (x > expRange) ? x : log(exp(x) + one);
      if (G == 1) y = x * (one - exp(-yy));
      if (G == 2) {
        scalar_t c = exp(-yy);
        y = x * c * (one - c);
      }
    }

    // swish
    if (A == 9) {
      if (G == 0)
        y = (x < -expRange) ? 0 : x / (exp(-x) + one);
      else {
        scalar_t c = exp(xref);
        scalar_t d = c + one;
        if (G == 1)
          y = (xref > halfExpRange) ? x : x * c * (xref + d) / (d * d);
        else
          y = (xref > halfExpRange)
                  ? 0
                  : x * c * (xref * (two - d) + two * d) / (d * d * d);
        yref = (xref < -expRange) ? 0 : xref / (exp(-xref) + one) * gain;
      }
    }

    // Apply gain.
    y *= gain * dy;

    // Clamp.
    if (clamp >= 0) {
      if (G == 0)
        y = (y > -clamp & y < clamp) ? y : (y >= 0) ? clamp : -clamp;
      else
        y = (yref > -clamp & yref < clamp) ? y : 0;
    }

    // Store.
    ((T *)p.y)[xi] = (T)y;
  }
}

//------------------------------------------------------------------------
// CUDA kernel selection.

template <class T>
void *choose_bias_act_kernel(const bias_act_kernel_params &p) {
  if (p.act == 1) return (void *)bias_act_kernel<T, 1>;
  if (p.act == 2) return (void *)bias_act_kernel<T, 2>;
  if (p.act == 3) return (void *)bias_act_kernel<T, 3>;
  if (p.act == 4) return (void *)bias_act_kernel<T, 4>;
  if (p.act == 5) return (void *)bias_act_kernel<T, 5>;
  if (p.act == 6) return (void *)bias_act_kernel<T, 6>;
  if (p.act == 7) return (void *)bias_act_kernel<T, 7>;
  if (p.act == 8) return (void *)bias_act_kernel<T, 8>;
  if (p.act == 9) return (void *)bias_act_kernel<T, 9>;
  return NULL;
}

//------------------------------------------------------------------------

static bool has_same_layout(torch::Tensor x, torch::Tensor y) {
  if (x.dim() != y.dim()) return false;
  for (int64_t i = 0; i < x.dim(); i++) {
    if (x.size(i) != y.size(i)) return false;
    if (x.size(i) >= 2 && x.stride(i) != y.stride(i)) return false;
  }
  return true;
}

//------------------------------------------------------------------------
torch::Tensor bias_act_op(const torch::Tensor &x, const torch::Tensor &b,
                          const torch::Tensor &xref, const torch::Tensor &yref,
                          const torch::Tensor &dy, int grad, int dim, int act,
                          float alpha, float gain, float clamp) {
  // Validate arguments.
  TORCH_CHECK(x.is_cuda(), "x must reside on CUDA device");
  TORCH_CHECK(
      b.numel() == 0 || (b.dtype() == x.dtype() && b.device() == x.device()),
      "b must have the same dtype and device as x");
  TORCH_CHECK(xref.numel() == 0 ||
                  (xref.sizes() == x.sizes() && xref.dtype() == x.dtype() &&
                   xref.device() == x.device()),
              "xref must have the same shape, dtype, and device as x");
  TORCH_CHECK(yref.numel() == 0 ||
                  (yref.sizes() == x.sizes() && yref.dtype() == x.dtype() &&
                   yref.device() == x.device()),
              "yref must have the same shape, dtype, and device as x");
  TORCH_CHECK(
      dy.numel() == 0 || (dy.sizes() == x.sizes() && dy.dtype() == x.dtype() &&
                          dy.device() == x.device()),
      "dy must have the same dtype and device as x");
  TORCH_CHECK(x.numel() <= INT_MAX, "x is too large");
  TORCH_CHECK(b.dim() == 1, "b must have rank 1");
  TORCH_CHECK(b.numel() == 0 || (dim >= 0 && dim < x.dim()),
              "dim is out of bounds");
  TORCH_CHECK(b.numel() == 0 || b.numel() == x.size(dim),
              "b has wrong number of elements");
  TORCH_CHECK(grad >= 0, "grad must be non-negative");

  // Validate layout.
  TORCH_CHECK(x.is_non_overlapping_and_dense(),
              "x must be non-overlapping and dense");
  TORCH_CHECK(b.is_contiguous(), "b must be contiguous");
  TORCH_CHECK(xref.numel() == 0 || has_same_layout(xref, x),
              "xref must have the same layout as x");
  TORCH_CHECK(yref.numel() == 0 || has_same_layout(yref, x),
              "yref must have the same layout as x");
  TORCH_CHECK(dy.numel() == 0 || has_same_layout(dy, x),
              "dy must have the same layout as x");

  // Create output tensor.
  const at::cuda::OptionalCUDAGuard device_guard(device_of(x));
  torch::Tensor y = torch::empty_like(x);
  TORCH_CHECK(has_same_layout(y, x), "y must have the same layout as x");

  // Initialize CUDA kernel parameters.
  bias_act_kernel_params p;
  p.x = x.data_ptr();
  p.b = (b.numel()) ? b.data_ptr() : NULL;
  p.xref = (xref.numel()) ? xref.data_ptr() : NULL;
  p.yref = (yref.numel()) ? yref.data_ptr() : NULL;
  p.dy = (dy.numel()) ? dy.data_ptr() : NULL;
  p.y = y.data_ptr();
  p.grad = grad;
  p.act = act;
  p.alpha = alpha;
  p.gain = gain;
  p.clamp = clamp;
  p.sizeX = (int)x.numel();
  p.sizeB = (int)b.numel();
  p.stepB = (b.numel()) ? (int)x.stride(dim) : 1;

  // Choose CUDA kernel.
  void *kernel;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "upfirdn2d_cuda", [&] {
    kernel = choose_bias_act_kernel<scalar_t>(p);
  });
  TORCH_CHECK(kernel, "no CUDA kernel found for the specified activation func");

  // Launch CUDA kernel.
  p.loopX = 4;
  int blockSize = 4 * 32;
  int gridSize = (p.sizeX - 1) / (p.loopX * blockSize) + 1;
  void *args[] = {&p};
#ifdef MMCV_WITH_HIP
  AT_CUDA_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(kernel), gridSize, blockSize, args, 0,
                                at::cuda::getCurrentCUDAStream()));
#else
  AT_CUDA_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(kernel), gridSize, blockSize, args, 0,
                                 at::cuda::getCurrentCUDAStream()));
#endif

  return y;
}
