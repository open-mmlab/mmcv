#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved

/*!
**************************************************************************************************
* InternImage
* Copyright (c) 2022 OpenGVLab
* Licensed under The MIT License [see LICENSE for details]
**************************************************************************************************
* Modified from
*https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/tree/pytorch_1.0.0
**************************************************************************************************
*/

#include <vector>
#include "dcnv3_cuda_kernal.cuh"
#include "pytorch_cuda_helper.hpp"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/torch.h>

at::Tensor dcnv3_cuda_forward(const at::Tensor &input, const at::Tensor &offset,
                              const at::Tensor &mask, const int kernel_h,
                              const int kernel_w, const int stride_h,
                              const int stride_w, const int pad_h,
                              const int pad_w, const int dilation_h,
                              const int dilation_w, const int group,
                              const int group_channels,
                              const float offset_scale, const int im2col_step) {
    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(offset.is_contiguous(), "offset tensor has to be contiguous");
    AT_ASSERTM(mask.is_contiguous(), "mask tensor has to be contiguous");
    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");

    const int batch = input.size(0);
    const int height_in = input.size(1);
    const int width_in = input.size(2);
    const int channels = input.size(3);
    const int height_out =
        (height_in + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h +
        1;
    const int width_out =
        (width_in + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w +
        1;
    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0,
               "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);
    AT_ASSERTM(
        channels == (group * group_channels),
        "Input channels and group times group channels wont match: (%d vs %d).",
        channels, group * group_channels);

    auto output =
        at::zeros({batch, height_out, width_out, group * group_channels},
                  input.options());

    const int batch_n = im2col_step_;
    auto output_n = output.view({batch / batch_n, batch_n, height_out,
                                 width_out, group * group_channels});
    auto per_input_size = height_in * width_in * group * group_channels;
    auto per_offset_size =
        height_out * width_out * group * kernel_h * kernel_w * 2;
    auto per_mask_size = height_out * width_out * group * kernel_h * kernel_w;
    for (int n = 0; n < batch / im2col_step_; ++n) {
        auto columns = output_n.select(0, n);
        // AT_DISPATCH_FLOATING_TYPES(
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            input.type(), "ms_deform_attn_forward_cuda", ([&] {
                dcnv3_im2col_cuda(
                    at::cuda::getCurrentCUDAStream(),
                    input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                    offset.data<scalar_t>() +
                        n * im2col_step_ * per_offset_size,
                    mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                    columns.data<scalar_t>(), kernel_h, kernel_w, stride_h,
                    stride_w, pad_h, pad_w, dilation_h, dilation_w, group,
                    group_channels, batch_n, height_in, width_in, height_out,
                    width_out, offset_scale);
            }));
    }

    return output;
}

std::vector<at::Tensor>
dcnv3_cuda_backward(const at::Tensor &input, const at::Tensor &offset,
                    const at::Tensor &mask, const int kernel_h,
                    const int kernel_w, const int stride_h, const int stride_w,
                    const int pad_h, const int pad_w, const int dilation_h,
                    const int dilation_w, const int group,
                    const int group_channels, const float offset_scale,
                    const at::Tensor &grad_output, const int im2col_step) {

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(offset.is_contiguous(), "offset tensor has to be contiguous");
    AT_ASSERTM(mask.is_contiguous(), "mask tensor has to be contiguous");
    AT_ASSERTM(grad_output.is_contiguous(),
               "grad_output tensor has to be contiguous");
    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");
    AT_ASSERTM(grad_output.type().is_cuda(),
               "grad_output must be a CUDA tensor");

    const int batch = input.size(0);
    const int height_in = input.size(1);
    const int width_in = input.size(2);
    const int channels = input.size(3);
    const int height_out =
        (height_in + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h +
        1;
    const int width_out =
        (width_in + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w +
        1;
    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0,
               "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);
    AT_ASSERTM(
        channels == (group * group_channels),
        "Input channels and group times group channels wont match: (%d vs %d).",
        channels, group * group_channels);

    auto dtype = input.dtype();
    if (dtype == at::kHalf) {
        dtype = at::kFloat;
    }

    auto grad_input = at::zeros_like(input, dtype);
    auto grad_offset = at::zeros_like(offset, dtype);
    auto grad_mask = at::zeros_like(mask, dtype);

    const int batch_n = im2col_step_;
    auto per_input_size = height_in * width_in * group * group_channels;
    auto per_offset_size =
        height_out * width_out * group * kernel_h * kernel_w * 2;
    auto per_mask_size = height_out * width_out * group * kernel_h * kernel_w;
    auto grad_output_n =
        grad_output.view({batch / im2col_step_, batch_n, height_out * width_out,
                          group, group_channels});

    for (int n = 0; n < batch / im2col_step_; ++n) {
        auto grad_output_g = grad_output_n.select(0, n);
        // AT_DISPATCH_FLOATING_TYPES(
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            input.type(), "ms_deform_attn_backward_cuda", ([&] {
                dcnv3_col2im_cuda(
                    at::cuda::getCurrentCUDAStream(),
                    grad_output_g.data<scalar_t>(),
                    input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                    offset.data<scalar_t>() +
                        n * im2col_step_ * per_offset_size,
                    mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                    kernel_h, kernel_w, stride_h, stride_w, pad_h, pad_w,
                    dilation_h, dilation_w, group, group_channels, batch_n,
                    height_in, width_in, height_out, width_out, offset_scale,
                    grad_input.data<opmath_t>() +
                        n * im2col_step_ * per_input_size,
                    grad_offset.data<opmath_t>() +
                        n * im2col_step_ * per_offset_size,
                    grad_mask.data<opmath_t>() +
                        n * im2col_step_ * per_mask_size);
            }));
    }

    if (input.dtype() == torch::kHalf) {
        return {grad_input.to(torch::kHalf), grad_offset.to(torch::kHalf),
                grad_mask.to(torch::kHalf)};
    } else {
        return {grad_input, grad_offset, grad_mask};
    }
}