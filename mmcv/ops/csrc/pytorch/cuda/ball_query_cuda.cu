// Copyright (c) OpenMMLab. All rights reserved
// Modified from
// https://github.com/sshaoshuai/Pointnet2.PyTorch/tree/master/pointnet2/src/ball_query_gpu.cu

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "ball_query_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void BallQueryForwardCUDAKernelLauncher(int b, int n, int m, float min_radius,
                                        float max_radius, int nsample,
                                        const Tensor new_xyz, const Tensor xyz,
                                        Tensor idx) {
  // new_xyz: (B, M, 3)
  // xyz: (B, N, 3)
  // output:
  //      idx: (B, M, nsample)

  at::cuda::CUDAGuard device_guard(new_xyz.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks(DIVUP(m, THREADS_PER_BLOCK),
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      new_xyz.scalar_type(), "ball_query_forward_cuda_kernel", [&] {
        ball_query_forward_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                b, n, m, min_radius, max_radius, nsample,
                new_xyz.data_ptr<scalar_t>(), xyz.data_ptr<scalar_t>(),
                idx.data_ptr<int>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}
