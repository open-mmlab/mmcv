#include "hip/hip_runtime.h"
// Copyright 2019 Yan Yan
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <ATen/ATen.h>
#include <utils/spconv/spconv/maxpool.h>
#include <utils/spconv/spconv/mp_helper.h>
#include <utils/spconv/tensorview/helper_launch.h>
#include <utils/spconv/tensorview/tensorview.h>

#include <chrono>
#include <limits>
#include <type_traits>
#include <utils/spconv/tensorview/helper_kernel.cuh>

#include "../spconv_utils.h"
#include "pytorch_cuda_helper.hpp"

template <typename scalar_t, typename Index, int NumTLP, int NumILP>
__global__ void maxPoolFwdBlockKernel(scalar_t *outFeatures,
                                      const scalar_t *inFeatures,
                                      const Index *indicesIn,
                                      const Index *indicesOut, int numHot,
                                      int numPlanes) {
  scalar_t in, out;
  int ILPStrideY[NumILP];
  Index idxo, idxi;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideY[ilp] = threadIdx.y + ilp * blockDim.y;
  outFeatures += blockIdx.y * NumTLP;
  inFeatures += blockIdx.y * NumTLP;
  for (int ix = blockIdx.x * blockDim.x; ix < numHot;
       ix += blockDim.x * gridDim.x) {
    {
#pragma unroll
      for (int ilp = 0; ilp < NumILP; ++ilp) {
        idxi = indicesIn[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
        idxo = indicesOut[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
        in = inFeatures[idxi];
        out = outFeatures[idxo];
        if (in > out) {
          outFeatures[idxo] = in;
        }
      }
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP>
__global__ void maxPoolFwdGenericBlockKernel(scalar_t *outFeatures,
                                             const scalar_t *inFeatures,
                                             const Index *indicesIn,
                                             const Index *indicesOut,
                                             int numHot, int numPlanes) {
  int ILPStrideX[NumILP];
  Index RI[NumILP];
  Index RO[NumILP];
  scalar_t in, out;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideX[ilp] = ilp * gridDim.x * blockDim.x;
  for (int ix : tv::KernelLoopX<int, NumILP>(numHot)) {
#pragma unroll
    for (int ilp = 0; ilp < NumILP; ilp++) {
      RI[ilp] = indicesIn[ix + ILPStrideX[ilp]] * numPlanes;
      RO[ilp] = indicesOut[ix + ILPStrideX[ilp]] * numPlanes;
    }
    for (int iy : tv::KernelLoopY<int>(numPlanes)) {
#pragma unroll
      for (int ilp = 0; ilp < NumILP; ++ilp) {
        in = inFeatures[RI[ilp] + iy];
        out = outFeatures[RO[ilp] + iy];
        if (in > out) {
          outFeatures[RO[ilp] + iy] = in;
        }
      }
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP,
          typename VecType>
__global__ void maxPoolFwdVecBlockKernel(scalar_t *outFeatures,
                                         const scalar_t *inFeatures,
                                         const Index *indicesIn,
                                         const Index *indicesOut, int numHot,
                                         int numPlanes) {
  int ILPStrideY[NumILP];
  constexpr int vecloadFactor = sizeof(VecType) / sizeof(scalar_t);
  scalar_t bufi[vecloadFactor];
  scalar_t bufo[vecloadFactor];
  Index idxi, idxo;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideY[ilp] = threadIdx.y + ilp * blockDim.y;
  outFeatures += blockIdx.y * NumTLP;
  inFeatures += blockIdx.y * NumTLP;
  for (int ix = blockIdx.x * blockDim.x * vecloadFactor; ix < numHot;
       ix += blockDim.x * gridDim.x * vecloadFactor) {
#pragma unroll
    for (int ilp = 0; ilp < NumILP; ++ilp) {
      idxi = indicesIn[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
      idxo = indicesOut[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
      reinterpret_cast<VecType *>(bufo)[0] =
          reinterpret_cast<VecType *>(outFeatures)[idxo];
      reinterpret_cast<VecType *>(bufi)[0] =
          reinterpret_cast<const VecType *>(inFeatures)[idxi];
#pragma unroll
      for (int i = 0; i < vecloadFactor; i++) {
        if (bufi[i] > bufo[i]) {
          bufo[i] = bufi[i];
        }
      }
      reinterpret_cast<VecType *>(outFeatures)[idxo] =
          reinterpret_cast<VecType *>(bufo)[0];
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP>
__global__ void maxPoolFwdGenericKernel(scalar_t *outFeatures,
                                        const scalar_t *inFeatures,
                                        const Index *indicesIn,
                                        const Index *indicesOut, int numHot,
                                        int numPlanes) {
  int ILPStrideX[NumILP];
  Index RI[NumILP];
  Index RO[NumILP];
  scalar_t in, out;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideX[ilp] = ilp * gridDim.x * blockDim.x;
  for (int ix : tv::KernelLoopX<int, NumILP>(numHot)) {
#pragma unroll
    for (int ilp = 0; ilp < NumILP; ilp++) {
      if (ix + ILPStrideX[ilp] < numHot) {
        RI[ilp] = indicesIn[ix + ILPStrideX[ilp]] * numPlanes;
        RO[ilp] = indicesOut[ix + ILPStrideX[ilp]] * numPlanes;
      }
    }
    for (int iy : tv::KernelLoopY<int>(numPlanes)) {
#pragma unroll
      for (int ilp = 0; ilp < NumILP; ++ilp) {
        if (ix + ILPStrideX[ilp] < numHot) {
          in = inFeatures[RI[ilp] + iy];
          out = outFeatures[RO[ilp] + iy];
          if (in > out) {
            outFeatures[RO[ilp] + iy] = in;
          }
        }
      }
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP>
__global__ void maxPoolBwdBlockKernel(const scalar_t *outFeatures,
                                      const scalar_t *inFeatures,
                                      const scalar_t *fout, scalar_t *fin,
                                      const Index *indicesIn,
                                      const Index *indicesOut, int numHot,
                                      int numPlanes) {
  scalar_t in, out;
  Index idxo, idxi;
  int ILPStrideY[NumILP];
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideY[ilp] = threadIdx.y + ilp * blockDim.y;
  outFeatures += blockIdx.y * NumTLP;
  inFeatures += blockIdx.y * NumTLP;
  fout += blockIdx.y * NumTLP;
  fin += blockIdx.y * NumTLP;
  for (int ix = blockIdx.x * blockDim.x; ix < numHot;
       ix += blockDim.x * gridDim.x) {
    {
#pragma unroll
      for (int ilp = 0; ilp < NumILP; ++ilp) {
        idxi = indicesIn[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
        idxo = indicesOut[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
        in = inFeatures[idxi];
        out = outFeatures[idxo];
        if (in == out) {
          fin[idxi] += fout[idxo];
        }
      }
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP>
__global__ void maxPoolBwdGenericBlockKernel(
    const scalar_t *outFeatures, const scalar_t *inFeatures,
    const scalar_t *fout, scalar_t *fin, const Index *indicesIn,
    const Index *indicesOut, int numHot, int numPlanes) {
  int ILPStrideX[NumILP];
  Index RI[NumILP];
  Index RO[NumILP];
  scalar_t in, out;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideX[ilp] = ilp * gridDim.x * blockDim.x;
  for (int ix : tv::KernelLoopX<int, NumILP>(numHot)) {
#pragma unroll
    for (int ilp = 0; ilp < NumILP; ilp++) {
      RI[ilp] = indicesIn[ix + ILPStrideX[ilp]] * numPlanes;
      RO[ilp] = indicesOut[ix + ILPStrideX[ilp]] * numPlanes;
    }
    for (int iy : tv::KernelLoopY<int>(numPlanes)) {
#pragma unroll
      for (int ilp = 0; ilp < NumILP; ++ilp) {
        in = inFeatures[RI[ilp] + iy];
        out = outFeatures[RO[ilp] + iy];
        if (in == out) {
          fin[RI[ilp] + iy] += fout[RO[ilp] + iy];
        }
      }
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP,
          typename VecType>
__global__ void maxPoolBwdVecBlockKernel(const scalar_t *outFeatures,
                                         const scalar_t *inFeatures,
                                         const scalar_t *fout, scalar_t *fin,
                                         const Index *indicesIn,
                                         const Index *indicesOut, int numHot,
                                         int numPlanes) {
  int ILPStrideY[NumILP];
  constexpr int vecloadFactor = sizeof(VecType) / sizeof(scalar_t);
  scalar_t bufi[vecloadFactor];
  scalar_t bufo[vecloadFactor];
  scalar_t bufdi[vecloadFactor];
  scalar_t bufdo[vecloadFactor];
  Index idxi, idxo;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideY[ilp] = threadIdx.y + ilp * blockDim.y;
  outFeatures += blockIdx.y * NumTLP;
  inFeatures += blockIdx.y * NumTLP;
  for (int ix = blockIdx.x * blockDim.x * vecloadFactor; ix < numHot;
       ix += blockDim.x * gridDim.x * vecloadFactor) {
#pragma unroll
    for (int ilp = 0; ilp < NumILP; ++ilp) {
      idxi = indicesIn[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
      idxo = indicesOut[ix + ILPStrideY[ilp]] * numPlanes + threadIdx.x;
      reinterpret_cast<VecType *>(bufo)[0] =
          reinterpret_cast<const VecType *>(outFeatures)[idxo];
      reinterpret_cast<VecType *>(bufi)[0] =
          reinterpret_cast<const VecType *>(inFeatures)[idxi];
      reinterpret_cast<VecType *>(bufdo)[0] =
          reinterpret_cast<const VecType *>(fout)[idxo];
      reinterpret_cast<VecType *>(bufdi)[0] =
          reinterpret_cast<VecType *>(fin)[idxi];

#pragma unroll
      for (int i = 0; i < vecloadFactor; i++) {
        if (bufi[i] == bufo[i]) {
          bufdi[i] += bufdo[i];
        }
      }
      reinterpret_cast<VecType *>(fin)[idxi] =
          reinterpret_cast<VecType *>(bufdi)[0];
    }
  }
}

template <typename scalar_t, typename Index, int NumTLP, int NumILP>
__global__ void maxPoolBwdGenericKernel(const scalar_t *outFeatures,
                                        const scalar_t *inFeatures,
                                        const scalar_t *fout, scalar_t *fin,
                                        const Index *indicesIn,
                                        const Index *indicesOut, int numHot,
                                        int numPlanes) {
  int ILPStrideX[NumILP];
  Index RI[NumILP];
  Index RO[NumILP];
  scalar_t in, out;
#pragma unroll
  for (int ilp = 0; ilp < NumILP; ilp++)
    ILPStrideX[ilp] = ilp * gridDim.x * blockDim.x;
  for (int ix : tv::KernelLoopX<int, NumILP>(numHot)) {
#pragma unroll
    for (int ilp = 0; ilp < NumILP; ilp++) {
      if (ix + ILPStrideX[ilp] < numHot) {
        RI[ilp] = indicesIn[ix + ILPStrideX[ilp]] * numPlanes;
        RO[ilp] = indicesOut[ix + ILPStrideX[ilp]] * numPlanes;
      }
    }
    for (int iy : tv::KernelLoopY<int>(numPlanes)) {
#pragma unroll
      for (int ilp = 0; ilp < NumILP; ++ilp) {
        if (ix + ILPStrideX[ilp] < numHot) {
          in = inFeatures[RI[ilp] + iy];
          out = outFeatures[RO[ilp] + iy];
          if (in == out) {
            fin[RI[ilp] + iy] += fout[RO[ilp] + iy];
          }
        }
      }
    }
  }
}

namespace functor {
template <typename scalar_t, typename Index>
struct SparseMaxPoolForwardFunctor<tv::TorchGPU, scalar_t, Index> {
  using vecload_type_t =
      std::conditional_t<std::is_same<scalar_t, at::Half>::value, int2, int4>;
  using kernel_block_t = mp_list_c<int, 64, 32, 16>;
  void operator()(const tv::TorchGPU &d, tv::TensorView<scalar_t> outFeatures,
                  tv::TensorView<const scalar_t> inFeatures,
                  tv::TensorView<const Index> indices, int size) {
    if (size <= 0) return;
    int numPlanes = inFeatures.dim(1);
    bool notFound = true;
    constexpr int vecloadFactor = sizeof(vecload_type_t) / sizeof(scalar_t);
    mp_for_each<kernel_block_t>([=, &outFeatures, &inFeatures, &indices,
                                 &notFound](auto NumTLP) {
      constexpr int NumILP = NumTLP / 4;

      int numHotBlock = (size / NumTLP) * NumTLP;
      if (notFound) {
        if (numPlanes % NumTLP == 0) {
          if (numHotBlock >= NumTLP) {
            maxPoolFwdVecBlockKernel<scalar_t, Index, int(NumTLP), NumILP,
                                     vecload_type_t>
                <<<dim3(std::min(size / NumTLP, 512), numPlanes / NumTLP),
                   dim3(NumTLP / vecloadFactor, NumTLP / NumILP), 0,
                   d.getStream()>>>(outFeatures.data(), inFeatures.data(),
                                    indices.subview(0).data(),
                                    indices.subview(1).data(), numHotBlock,
                                    numPlanes / vecloadFactor);
            TV_CHECK_CUDA_ERR();
          }

          if (size > numHotBlock) {
            maxPoolFwdGenericKernel<scalar_t, Index, int(NumTLP), NumILP>
                <<<dim3(1, numPlanes / NumTLP), dim3(NumTLP / NumILP, NumTLP),
                   0, d.getStream()>>>(outFeatures.data(), inFeatures.data(),
                                       indices.subview(0).data() + numHotBlock,
                                       indices.subview(1).data() + numHotBlock,
                                       size - numHotBlock, numPlanes);
            TV_CHECK_CUDA_ERR();
          }
          notFound = false;
        }
      }
    });

    if (notFound) {
      constexpr int NumTLP = 64;
      constexpr int NumILP = NumTLP / 4;
      int numHotBlock = (size / NumTLP) * NumTLP;
      if (numHotBlock >= NumTLP) {
        maxPoolFwdGenericBlockKernel<scalar_t, Index, NumTLP, NumILP>
            <<<dim3(size / NumTLP, tv::launch::DivUp(numPlanes, NumTLP)),
               dim3(NumTLP / NumILP, NumTLP), 0, d.getStream()>>>(
                outFeatures.data(), inFeatures.data(),
                indices.subview(0).data(), indices.subview(1).data(),
                numHotBlock, numPlanes);
        TV_CHECK_CUDA_ERR();
      }

      if (size > numHotBlock) {
        maxPoolFwdGenericKernel<scalar_t, Index, NumTLP, NumILP>
            <<<dim3(1, tv::launch::DivUp(numPlanes, NumTLP)),
               dim3(NumTLP / NumILP, NumTLP), 0, d.getStream()>>>(
                outFeatures.data(), inFeatures.data(),
                indices.subview(0).data() + numHotBlock,
                indices.subview(1).data() + numHotBlock, size - numHotBlock,
                numPlanes);
        TV_CHECK_CUDA_ERR();
      }
    }
  }
};

template <typename scalar_t, typename Index>
struct SparseMaxPoolBackwardFunctor<tv::TorchGPU, scalar_t, Index> {
  using vecload_type_t =
      std::conditional_t<std::is_same<scalar_t, at::Half>::value, int2, int4>;
  using kernel_block_t = mp_list_c<int, 64, 32, 16>;
  void operator()(const tv::TorchGPU &d,
                  tv::TensorView<const scalar_t> outFeatures,
                  tv::TensorView<const scalar_t> inFeatures,
                  tv::TensorView<const scalar_t> fout,
                  tv::TensorView<scalar_t> fin,
                  tv::TensorView<const Index> indices, int size) {
    if (size <= 0) return;
    int numPlanes = inFeatures.dim(1);
    bool notFound = true;
    constexpr int vecloadFactor = sizeof(vecload_type_t) / sizeof(scalar_t);
    mp_for_each<kernel_block_t>([=, &outFeatures, &inFeatures, &fout, &fin,
                                 &indices, &notFound](auto NumTLP) {
      constexpr int NumILP = NumTLP / 4;

      int numHotBlock = (size / NumTLP) * NumTLP;
      if (notFound) {
        if (numPlanes % NumTLP == 0) {
          if (numHotBlock >= NumTLP) {
            maxPoolBwdVecBlockKernel<scalar_t, Index, int(NumTLP), NumILP,
                                     vecload_type_t>
                <<<dim3(std::min(size / NumTLP, 512), numPlanes / NumTLP),
                   dim3(NumTLP / vecloadFactor, NumTLP / NumILP), 0,
                   d.getStream()>>>(outFeatures.data(), inFeatures.data(),
                                    fout.data(), fin.data(),
                                    indices.subview(0).data(),
                                    indices.subview(1).data(), numHotBlock,
                                    numPlanes / vecloadFactor);
            TV_CHECK_CUDA_ERR();
          }

          if (size > numHotBlock) {
            maxPoolBwdGenericKernel<scalar_t, Index, int(NumTLP), NumILP>
                <<<dim3(1, numPlanes / NumTLP), dim3(NumTLP / NumILP, NumTLP),
                   0, d.getStream()>>>(outFeatures.data(), inFeatures.data(),
                                       fout.data(), fin.data(),
                                       indices.subview(0).data() + numHotBlock,
                                       indices.subview(1).data() + numHotBlock,
                                       size - numHotBlock, numPlanes);
            TV_CHECK_CUDA_ERR();
          }
          notFound = false;
        }
      }
    });

    if (notFound) {
      constexpr int NumTLP = 64;
      constexpr int NumILP = NumTLP / 4;
      int numHotBlock = (size / NumTLP) * NumTLP;
      if (numHotBlock >= NumTLP) {
        maxPoolBwdGenericBlockKernel<scalar_t, Index, NumTLP, NumILP>
            <<<dim3(size / NumTLP, tv::launch::DivUp(numPlanes, NumTLP)),
               dim3(NumTLP / NumILP, NumTLP), 0, d.getStream()>>>(
                outFeatures.data(), inFeatures.data(), fout.data(), fin.data(),
                indices.subview(0).data(), indices.subview(1).data(),
                numHotBlock, numPlanes);
        TV_CHECK_CUDA_ERR();
      }

      if (size > numHotBlock) {
        maxPoolBwdGenericKernel<scalar_t, Index, NumTLP, NumILP>
            <<<dim3(1, tv::launch::DivUp(numPlanes, NumTLP)),
               dim3(NumTLP / NumILP, NumTLP), 0, d.getStream()>>>(
                outFeatures.data(), inFeatures.data(), fout.data(), fin.data(),
                indices.subview(0).data() + numHotBlock,
                indices.subview(1).data() + numHotBlock, size - numHotBlock,
                numPlanes);
        TV_CHECK_CUDA_ERR();
      }
    }
  }
};

}  // namespace functor

#define DECLARE_GPU_SPECS_T_INDEX(scalar_t, Index)                             \
  template struct functor::SparseMaxPoolForwardFunctor<tv::TorchGPU, scalar_t, \
                                                       Index>;                 \
  template struct functor::SparseMaxPoolBackwardFunctor<tv::TorchGPU,          \
                                                        scalar_t, Index>;

#define DECLARE_GPU_SPECS(scalar_t) DECLARE_GPU_SPECS_T_INDEX(scalar_t, int);

DECLARE_GPU_SPECS(float);
DECLARE_GPU_SPECS(double);
DECLARE_GPU_SPECS(at::Half);

#undef DECLARE_GPU_SPECS
#undef DECLARE_GPU_SPECS_T_INDEX
