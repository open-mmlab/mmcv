#include "hip/hip_runtime.h"
/*!
**************************************************************************************************
* Deformable DETR
* Copyright (c) 2020 SenseTime. All Rights Reserved.
* Licensed under the Apache License, Version 2.0 [see LICENSE for details]
**************************************************************************************************
* Modified from
*https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/tree/pytorch_1.0.0
**************************************************************************************************
*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <THC/THCAtomics.cuh>
#include <vector>

#include "ms_deform_attn_cuda_kernel.cuh"

template <typename scalar_t>
void ms_deformable_im2col_cuda(hipStream_t stream, const scalar_t *data_value,
                               const int64_t *data_spatial_shapes,
                               const int64_t *data_level_start_index,
                               const scalar_t *data_sampling_loc,
                               const scalar_t *data_attn_weight,
                               const int batch_size, const int spatial_size,
                               const int num_heads, const int channels,
                               const int num_levels, const int num_query,
                               const int num_point, scalar_t *data_col) {
  const int num_kernels = batch_size * num_query * num_heads * channels;
  const int num_actual_kernels = batch_size * num_query * num_heads * channels;
  const int num_threads = CUDA_NUM_THREADS;
  ms_deformable_im2col_gpu_kernel<scalar_t>
      <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0, stream>>>(
          num_kernels, data_value, data_spatial_shapes, data_level_start_index,
          data_sampling_loc, data_attn_weight, batch_size, spatial_size,
          num_heads, channels, num_levels, num_query, num_point, data_col);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in ms_deformable_im2col_cuda: %s\n", hipGetErrorString(err));
  }
}

template <typename scalar_t>
void ms_deformable_col2im_cuda(
    hipStream_t stream, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int num_threads =
      (channels > CUDA_NUM_THREADS) ? CUDA_NUM_THREADS : channels;
  const int num_kernels = batch_size * num_query * num_heads * channels;
  const int num_actual_kernels = batch_size * num_query * num_heads * channels;
  if (channels > 1024) {
    if ((channels & 1023) == 0) {
      ms_deformable_col2im_gpu_kernel_shm_reduce_v2_multi_blocks<scalar_t>
          <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads,
             num_threads * 3 * sizeof(scalar_t), stream>>>(
              num_kernels, grad_col, data_value, data_spatial_shapes,
              data_level_start_index, data_sampling_loc, data_attn_weight,
              batch_size, spatial_size, num_heads, channels, num_levels,
              num_query, num_point, grad_value, grad_sampling_loc,
              grad_attn_weight);
    } else {
      ms_deformable_col2im_gpu_kernel_gm<scalar_t>
          <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
             stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                       data_level_start_index, data_sampling_loc,
                       data_attn_weight, batch_size, spatial_size, num_heads,
                       channels, num_levels, num_query, num_point, grad_value,
                       grad_sampling_loc, grad_attn_weight);
    }
  } else {
    switch (channels) {
      case 1:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t,
                                                                      1>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 2:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t,
                                                                      2>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 4:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t,
                                                                      4>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 8:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t,
                                                                      8>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 16:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t,
                                                                      16>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 32:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t,
                                                                      32>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 64:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t,
                                                                      64>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 128:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t,
                                                                      128>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 256:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t,
                                                                      256>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 512:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t,
                                                                      512>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 1024:
        ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t,
                                                                      1024>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      default:
        if (channels < 64) {
          ms_deformable_col2im_gpu_kernel_shm_reduce_v1<scalar_t>
              <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads,
                 num_threads * 3 * sizeof(scalar_t), stream>>>(
                  num_kernels, grad_col, data_value, data_spatial_shapes,
                  data_level_start_index, data_sampling_loc, data_attn_weight,
                  batch_size, spatial_size, num_heads, channels, num_levels,
                  num_query, num_point, grad_value, grad_sampling_loc,
                  grad_attn_weight);
        } else {
          ms_deformable_col2im_gpu_kernel_shm_reduce_v2<scalar_t>
              <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads,
                 num_threads * 3 * sizeof(scalar_t), stream>>>(
                  num_kernels, grad_col, data_value, data_spatial_shapes,
                  data_level_start_index, data_sampling_loc, data_attn_weight,
                  batch_size, spatial_size, num_heads, channels, num_levels,
                  num_query, num_point, grad_value, grad_sampling_loc,
                  grad_attn_weight);
        }
    }
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in ms_deformable_col2im_cuda: %s\n", hipGetErrorString(err));
  }
}

at::Tensor ms_deform_attn_cuda_forward(const at::Tensor &value,
                                       const at::Tensor &spatial_shapes,
                                       const at::Tensor &level_start_index,
                                       const at::Tensor &sampling_loc,
                                       const at::Tensor &attn_weight,
                                       const int im2col_step) {
  AT_ASSERTM(value.is_contiguous(), "value tensor has to be contiguous");
  AT_ASSERTM(spatial_shapes.is_contiguous(),
             "spatial_shapes tensor has to be contiguous");
  AT_ASSERTM(level_start_index.is_contiguous(),
             "level_start_index tensor has to be contiguous");
  AT_ASSERTM(sampling_loc.is_contiguous(),
             "sampling_loc tensor has to be contiguous");
  AT_ASSERTM(attn_weight.is_contiguous(),
             "attn_weight tensor has to be contiguous");

  AT_ASSERTM(value.type().is_cuda(), "value must be a CUDA tensor");
  AT_ASSERTM(spatial_shapes.type().is_cuda(),
             "spatial_shapes must be a CUDA tensor");
  AT_ASSERTM(level_start_index.type().is_cuda(),
             "level_start_index must be a CUDA tensor");
  AT_ASSERTM(sampling_loc.type().is_cuda(),
             "sampling_loc must be a CUDA tensor");
  AT_ASSERTM(attn_weight.type().is_cuda(), "attn_weight must be a CUDA tensor");

  const int batch = value.size(0);
  const int spatial_size = value.size(1);
  const int num_heads = value.size(2);
  const int channels = value.size(3);

  const int num_levels = spatial_shapes.size(0);

  const int num_query = sampling_loc.size(1);
  const int num_point = sampling_loc.size(4);

  const int im2col_step_ = std::min(batch, im2col_step);

  AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)",
             batch, im2col_step_);

  auto output =
      at::zeros({batch, num_query, num_heads, channels}, value.options());

  const int batch_n = im2col_step_;
  auto output_n = output.view(
      {batch / im2col_step_, batch_n, num_query, num_heads, channels});
  auto per_value_size = spatial_size * num_heads * channels;
  auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
  auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
  for (int n = 0; n < batch / im2col_step_; ++n) {
    auto columns = output_n.select(0, n);
    AT_DISPATCH_FLOATING_TYPES(
        value.type(), "ms_deform_attn_forward_cuda", ([&] {
          ms_deformable_im2col_cuda(
              at::cuda::getCurrentCUDAStream(),
              value.data<scalar_t>() + n * im2col_step_ * per_value_size,
              spatial_shapes.data<int64_t>(), level_start_index.data<int64_t>(),
              sampling_loc.data<scalar_t>() +
                  n * im2col_step_ * per_sample_loc_size,
              attn_weight.data<scalar_t>() +
                  n * im2col_step_ * per_attn_weight_size,
              batch_n, spatial_size, num_heads, channels, num_levels, num_query,
              num_point, columns.data<scalar_t>());
        }));
  }

  output = output.view({batch, num_query, num_heads * channels});

  return output;
}

void ms_deform_attn_cuda_backward(
    const at::Tensor &value, const at::Tensor &spatial_shapes,
    const at::Tensor &level_start_index, const at::Tensor &sampling_loc,
    const at::Tensor &attn_weight, const at::Tensor &grad_output,
    at::Tensor &grad_value, at::Tensor &grad_sampling_loc,
    at::Tensor &grad_attn_weight, const int im2col_step) {
  AT_ASSERTM(value.is_contiguous(), "value tensor has to be contiguous");
  AT_ASSERTM(spatial_shapes.is_contiguous(),
             "spatial_shapes tensor has to be contiguous");
  AT_ASSERTM(level_start_index.is_contiguous(),
             "level_start_index tensor has to be contiguous");
  AT_ASSERTM(sampling_loc.is_contiguous(),
             "sampling_loc tensor has to be contiguous");
  AT_ASSERTM(attn_weight.is_contiguous(),
             "attn_weight tensor has to be contiguous");
  AT_ASSERTM(grad_output.is_contiguous(),
             "grad_output tensor has to be contiguous");

  AT_ASSERTM(value.type().is_cuda(), "value must be a CUDA tensor");
  AT_ASSERTM(spatial_shapes.type().is_cuda(),
             "spatial_shapes must be a CUDA tensor");
  AT_ASSERTM(level_start_index.type().is_cuda(),
             "level_start_index must be a CUDA tensor");
  AT_ASSERTM(sampling_loc.type().is_cuda(),
             "sampling_loc must be a CUDA tensor");
  AT_ASSERTM(attn_weight.type().is_cuda(), "attn_weight must be a CUDA tensor");
  AT_ASSERTM(grad_output.type().is_cuda(), "grad_output must be a CUDA tensor");

  const int batch = value.size(0);
  const int spatial_size = value.size(1);
  const int num_heads = value.size(2);
  const int channels = value.size(3);

  const int num_levels = spatial_shapes.size(0);

  const int num_query = sampling_loc.size(1);
  const int num_point = sampling_loc.size(4);

  const int im2col_step_ = std::min(batch, im2col_step);

  AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)",
             batch, im2col_step_);

  const int batch_n = im2col_step_;
  auto per_value_size = spatial_size * num_heads * channels;
  auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
  auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
  auto grad_output_n = grad_output.view(
      {batch / im2col_step_, batch_n, num_query, num_heads, channels});

  for (int n = 0; n < batch / im2col_step_; ++n) {
    auto grad_output_g = grad_output_n.select(0, n);
    AT_DISPATCH_FLOATING_TYPES(
        value.type(), "ms_deform_attn_backward_cuda", ([&] {
          ms_deformable_col2im_cuda(
              at::cuda::getCurrentCUDAStream(), grad_output_g.data<scalar_t>(),
              value.data<scalar_t>() + n * im2col_step_ * per_value_size,
              spatial_shapes.data<int64_t>(), level_start_index.data<int64_t>(),
              sampling_loc.data<scalar_t>() +
                  n * im2col_step_ * per_sample_loc_size,
              attn_weight.data<scalar_t>() +
                  n * im2col_step_ * per_attn_weight_size,
              batch_n, spatial_size, num_heads, channels, num_levels, num_query,
              num_point,
              grad_value.data<scalar_t>() + n * im2col_step_ * per_value_size,
              grad_sampling_loc.data<scalar_t>() +
                  n * im2col_step_ * per_sample_loc_size,
              grad_attn_weight.data<scalar_t>() +
                  n * im2col_step_ * per_attn_weight_size);
        }));
  }
}
