#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved
#include "bbox_overlaps_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

// Disable fp16 on ROCm device
#ifndef HIP_DIFF
#if __CUDA_ARCH__ >= 530
template <>
__global__ void bbox_overlaps_cuda_kernel<at::Half>(
    const at::Half* bbox1, const at::Half* bbox2, at::Half* ious,
    const int num_bbox1, const int num_bbox2, const int mode,
    const bool aligned, const int offset) {
  bbox_overlaps_cuda_kernel_half(reinterpret_cast<const __half*>(bbox1),
                                 reinterpret_cast<const __half*>(bbox2),
                                 reinterpret_cast<__half*>(ious), num_bbox1,
                                 num_bbox2, mode, aligned, offset);
}
#endif  // __CUDA_ARCH__ >= 530
#endif  // HIP_DIFF

void BBoxOverlapsCUDAKernelLauncher(const Tensor bboxes1, const Tensor bboxes2,
                                    Tensor ious, const int mode,
                                    const bool aligned, const int offset) {
  int output_size = ious.numel();
  int num_bbox1 = bboxes1.size(0);
  int num_bbox2 = bboxes2.size(0);

  at::cuda::CUDAGuard device_guard(bboxes1.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bboxes1.scalar_type(), "bbox_overlaps_cuda_kernel", ([&] {
        bbox_overlaps_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                bboxes1.data_ptr<scalar_t>(), bboxes2.data_ptr<scalar_t>(),
                ious.data_ptr<scalar_t>(), num_bbox1, num_bbox2, mode, aligned,
                offset);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
