// Copyright (c) OpenMMLab. All rights reserved
#include "pytorch_cuda_helper.hpp"
#include "roi_align_rotated_cuda_kernel.cuh"

void ROIAlignRotatedForwardCUDAKernelLauncher(
    const at::Tensor features, const at::Tensor rois, const float spatial_scale,
    const int sample_num, const bool aligned, const bool clockwise,
    const int channels, const int height, const int width, const int num_rois,
    const int pooled_height, const int pooled_width, at::Tensor output) {
  const int output_size = num_rois * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.type(), "ROIAlignRotatedLaucherForward", ([&] {
        const scalar_t *bottom_data = features.data<scalar_t>();
        const scalar_t *rois_data = rois.data<scalar_t>();
        scalar_t *top_data = output.data<scalar_t>();

        roi_align_rotated_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data, rois_data, scalar_t(spatial_scale),
                sample_num, aligned, clockwise, channels, height, width,
                pooled_height, pooled_width, top_data);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void ROIAlignRotatedBackwardCUDAKernelLauncher(
    const at::Tensor top_grad, const at::Tensor rois, const float spatial_scale,
    const int sample_num, const bool aligned, const bool clockwise,
    const int channels, const int height, const int width, const int num_rois,
    const int pooled_height, const int pooled_width, at::Tensor bottom_grad) {
  const int output_size = num_rois * pooled_height * pooled_width * channels;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.type(), "ROIAlignLaucherBackward", ([&] {
        const scalar_t *top_diff = top_grad.data<scalar_t>();
        const scalar_t *rois_data = rois.data<scalar_t>();
        scalar_t *bottom_diff = bottom_grad.data<scalar_t>();
        roi_align_rotated_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, top_diff, rois_data, spatial_scale, sample_num,
                aligned, clockwise, channels, height, width, pooled_height,
                pooled_width, bottom_diff);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
