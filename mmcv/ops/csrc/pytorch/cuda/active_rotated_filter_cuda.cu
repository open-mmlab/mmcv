// Copyright (c) OpenMMLab. All rights reserved.
// Modified from
// https://github.com/csuhan/s2anet/blob/master/mmdet/ops/orn/src/cuda/ActiveRotatingFilter_cuda.cu
#include "active_rotated_filter_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void ARFForwardLauncher(const Tensor input, const Tensor indices,
                        Tensor output) {
  int nOutputPlane = input.size(0);
  int nInputPlane = input.size(1);
  int num_orientations = input.size(2);
  int kH = input.size(3);
  int kW = input.size(4);
  int num_rotations = indices.size(3);
  int nEntry = num_orientations * kH * kW;
  int output_size = output.numel();

  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "ARF_forward", [&] {
    ARF_forward_cuda_kernel<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
            output_size, input.data_ptr<scalar_t>(),
            indices.data_ptr<int>(), nInputPlane, nOutputPlane,
            num_orientations, num_rotations, nEntry,
            output.data<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
}

void ARFBackwardLauncher(const Tensor grad_out, const Tensor indices,
                         Tensor grad_in) {
  int num_orientations = indices.size(0);
  int kH = indices.size(1);
  int kW = indices.size(2);
  int num_rotations = indices.size(3);
  int nOutputPlane = grad_out.size(0) / num_rotations;
  int nInputPlane = grad_out.size(1) / num_orientations;
  int nEntry = num_orientations * kH * kW;
  int output_size = grad_in.numel();

  at::cuda::CUDAGuard device_guard(indices.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(grad_out.scalar_type(), "ARF_backward", [&] {
    ARF_backward_cuda_kernel<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
            output_size, grad_out.data_ptr<scalar_t>(),
            indices.data_ptr<int>(), nInputPlane, nOutputPlane,
            num_orientations, num_rotations, nEntry,
            grad_in.data_ptr<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
}
