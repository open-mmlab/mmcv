#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved
#include "pytorch_cuda_helper.hpp"
#include "sigmoid_focal_loss_cuda_kernel.cuh"
#include "softmax_focal_loss_cuda_kernel.cuh"

void SigmoidFocalLossForwardCUDAKernelLauncher(Tensor input, Tensor target,
                                               Tensor weight, Tensor output,
                                               const float gamma,
                                               const float alpha) {
  int output_size = output.numel();
  int num_classes = input.size(1);
  AT_ASSERTM(target.max().item<int64_t>() <= (int64_t)num_classes,
             "target label should smaller or equal than num classes");
  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "sigmoid_focal_loss_forward_cuda_kernel", [&] {
        sigmoid_focal_loss_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, input.data_ptr<scalar_t>(),
                target.data_ptr<int64_t>(), weight.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(), gamma, alpha, num_classes);
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void SigmoidFocalLossBackwardCUDAKernelLauncher(Tensor input, Tensor target,
                                                Tensor weight,
                                                Tensor grad_input,
                                                const float gamma,
                                                const float alpha) {
  int output_size = grad_input.numel();
  int num_classes = input.size(1);

  at::cuda::CUDAGuard device_guard(grad_input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "sigmoid_focal_loss_backward_cuda_kernel", [&] {
        sigmoid_focal_loss_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, input.data_ptr<scalar_t>(),
                target.data_ptr<int64_t>(), weight.data_ptr<scalar_t>(),
                grad_input.data_ptr<scalar_t>(), gamma, alpha, num_classes);
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void SoftmaxFocalLossForwardCUDAKernelLauncher(const Tensor input, const Tensor target,
                                               const Tensor weight, Tensor output,
                                               Tensor log_softmax_prob,
                                               const float gamma,
                                               const float alpha) {
  int output_size = output.numel();
  int num_classes = input.size(1);

  AT_ASSERTM(target.max().item<int64_t>() <= (int64_t)num_classes,
             "target label should smaller or equal than num classes");
  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "softmax_focal_loss_forward_cuda_kernel", [&] {
        softmax_focal_loss_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size,
                input.data_ptr<scalar_t>(), target.data_ptr<int64_t>(),
                weight.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                log_softmax_prob.data_ptr<scalar_t>(),
                gamma, alpha, num_classes);
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void SoftmaxFocalLossBackwardCUDAKernelLauncher(const Tensor log_softmax_prob,
                                                const Tensor target,
                                                const Tensor weight,
                                                Tensor sum_buff_along_class,
                                                Tensor grad_input,
                                                const float gamma,
                                                const float alpha) {
  int output_size = grad_input.numel();
  int num_classes = log_softmax_prob.size(1);

  at::cuda::CUDAGuard device_guard(grad_input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      log_softmax_prob.scalar_type(), "softmax_focal_loss_backward_cuda_kernel", [&] {
        softmax_focal_loss_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size,
                log_softmax_prob.data_ptr<scalar_t>(), target.data_ptr<int64_t>(),
                weight.data_ptr<scalar_t>(), sum_buff_along_class.data_ptr<scalar_t>(),
                grad_input.data_ptr<scalar_t>(),
                gamma, alpha, num_classes);
      });

  AT_CUDA_CHECK(hipGetLastError());
}
