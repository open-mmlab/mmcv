// Copyright (c) OpenMMLab. All rights reserved
#include "carafe_naive_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"
#include "pytorch_device_registry.hpp"

void CARAFENAIVEForwardCUDAKernelLauncher(const Tensor features,
                                          const Tensor masks, Tensor output,
                                          const int kernel_size,
                                          const int group_size,
                                          const int scale_factor) {
  int output_size = output.numel();
  int channels = output.size(1);
  int height = output.size(2);
  int width = output.size(3);

  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "CARAFENAIVEForward", ([&] {
        carafe_naive_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, features.data_ptr<scalar_t>(),
                masks.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                kernel_size, group_size, scale_factor, channels, height, width);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void CARAFENAIVEBackwardCUDAKernelLauncher(
    const Tensor top_grad, const Tensor features, const Tensor masks,
    Tensor bottom_grad, Tensor mask_grad, const int kernel_size,
    const int group_size, const int scale_factor) {
  int output_size = top_grad.numel();
  int channels = top_grad.size(1);
  int height = top_grad.size(2);
  int width = top_grad.size(3);

  at::cuda::CUDAGuard device_guard(top_grad.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "CARAFENAIVEBackward", ([&] {
        carafe_naive_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, top_grad.data_ptr<scalar_t>(),
                features.data_ptr<scalar_t>(), masks.data_ptr<scalar_t>(),
                bottom_grad.data_ptr<scalar_t>(),
                mask_grad.data_ptr<scalar_t>(), kernel_size, group_size,
                scale_factor, channels, height, width);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void carafe_naive_forward_cuda(Tensor features, Tensor masks, Tensor output,
                               int kernel_size, int group_size,
                               int scale_factor) {
  CARAFENAIVEForwardCUDAKernelLauncher(features, masks, output, kernel_size,
                                       group_size, scale_factor);
}

void carafe_naive_backward_cuda(Tensor top_grad, Tensor features, Tensor masks,
                                Tensor bottom_grad, Tensor mask_grad,
                                int kernel_size, int group_size,
                                int scale_factor) {
  CARAFENAIVEBackwardCUDAKernelLauncher(top_grad, features, masks, bottom_grad,
                                        mask_grad, kernel_size, group_size,
                                        scale_factor);
}
void carafe_naive_forward_impl(Tensor features, Tensor masks, Tensor output,
                               int kernel_size, int group_size,
                               int scale_factor);

void carafe_naive_backward_impl(Tensor top_grad, Tensor features, Tensor masks,
                                Tensor bottom_grad, Tensor mask_grad,
                                int kernel_size, int group_size,
                                int scale_factor);

REGISTER_DEVICE_IMPL(carafe_naive_forward_impl, CUDA,
                     carafe_naive_forward_cuda);
REGISTER_DEVICE_IMPL(carafe_naive_backward_impl, CUDA,
                     carafe_naive_backward_cuda);
