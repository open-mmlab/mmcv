// Copyright (c) OpenMMLab. All rights reserved
// modified from
// https://github.com/SDL-GuoZonghao/BeyondBoundingBox/blob/main/mmdet/ops/minareabbox/src/minareabbox_kernel.cu
#include "min_area_polygons_cuda.cuh"
#include "pytorch_cuda_helper.hpp"

void MinAreaPolygonsCUDAKernelLauncher(const Tensor pointsets,
                                       Tensor polygons) {
  int num_pointsets = pointsets.size(0);
  const int output_size = polygons.numel();
  at::cuda::CUDAGuard device_guard(pointsets.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      pointsets.scalar_type(), "min_area_polygons_cuda_kernel", ([&] {
        min_area_polygons_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                num_pointsets, pointsets.data_ptr<scalar_t>(),
                polygons.data_ptr<scalar_t>());
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
