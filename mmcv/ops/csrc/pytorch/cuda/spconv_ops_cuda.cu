#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <torch/script.h>
#include <utils/spconv/spconv/indice.h>
#include <utils/spconv/spconv/reordering.h>

#include "../spconv_utils.h"
#include "pytorch_cuda_helper.hpp"

template <unsigned NDim>
std::vector<torch::Tensor> GetIndicePairsForwardCUDAKernelLauncher(
    torch::Tensor indices, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose) {
  at::cuda::CUDAGuard device_guard(indices.device());
  bool subM = _subM != 0;
  bool transpose = _transpose != 0;
  auto numAct = indices.size(0);
  auto coorDim = indices.size(1) - 1;
  TV_ASSERT_RT_ERR(NDim == coorDim, "error");
  TV_ASSERT_RT_ERR(kernelSize.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(outSpatialShape.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(stride.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(padding.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(outPadding.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(dilation.size() == coorDim, "error");
  auto kernelVolume = kernelSize[0];
  for (int i = 1; i < kernelSize.size(); ++i) {
    kernelVolume *= kernelSize[i];
  }
  TV_ASSERT_RT_ERR(kernelVolume <= 4096, "error");
  auto outputVolume = outSpatialShape[0];
  for (int i = 1; i < outSpatialShape.size(); ++i) {
    outputVolume *= outSpatialShape[i];
  }
  torch::Tensor indicePairs =
      torch::full({kernelVolume, 2, numAct}, -1,
                  torch::dtype(torch::kInt32).device(indices.device()));
  torch::Tensor indiceNum = torch::zeros(
      {kernelVolume}, torch::dtype(torch::kInt32).device(indices.device()));
  torch::Tensor gridOut =
      torch::full({batchSize * outputVolume}, -1,
                  torch::dtype(torch::kInt32).device(indices.device()));
  int64_t numActOut = -1;
  tv::SimpleVector<int, NDim> outSpatialShape32;
  tv::SimpleVector<int, NDim> kernelSize32;
  tv::SimpleVector<int, NDim> stride32;
  tv::SimpleVector<int, NDim> padding32;
  tv::SimpleVector<int, NDim> dilation32;
  auto indicePairUnique = torch::full(
      {indicePairs.numel() / 2 + 1}, std::numeric_limits<int>::max(),
      torch::dtype(torch::kInt32).device(indices.device()));
  for (int i = 0; i < NDim; ++i) {
    outSpatialShape32.push_back(outSpatialShape[i]);
    kernelSize32.push_back(kernelSize[i]);
    if (subM) {
      stride32.push_back(1);
      padding32.push_back(kernelSize[i] / 2);
      dilation32.push_back(dilation[i]);
    } else {
      stride32.push_back(stride[i]);
      padding32.push_back(padding[i]);
      dilation32.push_back(dilation[i]);
    }
  }
  if (subM) {
    if (indices.device().type() == torch::kCPU) {
      auto getIndicePairFtor =
          functor::CreateSubMIndicePairFunctor<tv::CPU, int, int, NDim>();
      numActOut = getIndicePairFtor(
          tv::CPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(gridOut), tv::torch2tv<int>(indicePairs),
          tv::torch2tv<int>(indiceNum), kernelSize32, stride32, padding32,
          dilation32, outSpatialShape32, transpose);
    } else {
      auto getIndicePairFtor =
          functor::CreateSubMIndicePairFunctor<tv::TorchGPU, int, int, NDim>();
      numActOut = getIndicePairFtor(
          tv::TorchGPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(gridOut), tv::torch2tv<int>(indicePairs),
          tv::torch2tv<int>(indiceNum), kernelSize32, stride32, padding32,
          dilation32, outSpatialShape32, transpose);
    }
    return {indices, indicePairs, indiceNum};
  } else {
    torch::Tensor outInds =
        torch::zeros({numAct * kernelVolume, coorDim + 1},
                     torch::dtype(torch::kInt32).device(indices.device()));
    if (indices.device().type() == torch::kCPU) {
      auto getIndicePairFtor =
          functor::CreateConvIndicePairFunctor<tv::CPU, int, int, NDim>();
      numActOut = getIndicePairFtor(
          tv::CPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(outInds), tv::torch2tv<int>(gridOut),
          tv::torch2tv<int>(indicePairs), tv::torch2tv<int>(indiceNum),
          kernelSize32, stride32, padding32, dilation32, outSpatialShape32,
          transpose);
    } else {
      auto getIndicePairFtorP1 =
          functor::CreateConvIndicePairFunctorP1<tv::TorchGPU, int, int,
                                                 NDim>();
      auto getIndicePairFtorP2 =
          functor::CreateConvIndicePairFunctorP2<tv::TorchGPU, int, int,
                                                 NDim>();
      numActOut = getIndicePairFtorP1(
          tv::TorchGPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(outInds), tv::torch2tv<int>(gridOut),
          tv::torch2tv<int>(indicePairs), tv::torch2tv<int>(indiceNum),
          tv::torch2tv<int>(indicePairUnique), kernelSize32, stride32,
          padding32, dilation32, outSpatialShape32, transpose);
      if (numActOut > 0) {
        auto res = torch::_unique(indicePairUnique);
        indicePairUnique = std::get<0>(res);
        numActOut = getIndicePairFtorP2(
            tv::TorchGPU(), tv::torch2tv<const int>(indices),
            tv::torch2tv<int>(outInds), tv::torch2tv<int>(gridOut),
            tv::torch2tv<int>(indicePairs), tv::torch2tv<int>(indiceNum),
            tv::torch2tv<int>(indicePairUnique), outSpatialShape32, transpose);
      }
    }
    return {outInds.slice(0, 0, numActOut), indicePairs, indiceNum};
  }
}

template <unsigned NDim>
std::vector<torch::Tensor> GetIndicePairsBackwardCUDAKernelLauncher(
    torch::Tensor indices, torch::Tensor gridOut, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose) {
  at::cuda::CUDAGuard device_guard(indices.device());
  bool subM = _subM != 0;
  bool transpose = _transpose != 0;
  auto numAct = indices.size(0);
  auto coorDim = indices.size(1) - 1;
  TV_ASSERT_RT_ERR(NDim == coorDim, "error");
  TV_ASSERT_RT_ERR(kernelSize.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(outSpatialShape.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(stride.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(padding.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(outPadding.size() == coorDim, "error");
  TV_ASSERT_RT_ERR(dilation.size() == coorDim, "error");
  auto kernelVolume = kernelSize[0];
  for (int i = 1; i < kernelSize.size(); ++i) {
    kernelVolume *= kernelSize[i];
  }
  TV_ASSERT_RT_ERR(kernelVolume <= 4096, "error");
  auto outputVolume = outSpatialShape[0];
  for (int i = 1; i < outSpatialShape.size(); ++i) {
    outputVolume *= outSpatialShape[i];
  }
  TV_ASSERT_INVALID_ARG(gridOut.numel() >= outputVolume * batchSize, "error");
  torch::Tensor indicePairs =
      torch::full({kernelVolume, 2, numAct}, -1,
                  torch::dtype(torch::kInt32).device(indices.device()));
  torch::Tensor indiceNum = torch::zeros(
      {kernelVolume}, torch::dtype(torch::kInt32).device(indices.device()));
  int64_t numActOut = -1;
  tv::SimpleVector<int, NDim> outSpatialShape32;
  tv::SimpleVector<int, NDim> kernelSize32;
  tv::SimpleVector<int, NDim> stride32;
  tv::SimpleVector<int, NDim> padding32;
  tv::SimpleVector<int, NDim> dilation32;
  auto indicePairUnique = torch::full(
      {indicePairs.numel() / 2 + 1}, std::numeric_limits<int>::max(),
      torch::dtype(torch::kInt32).device(indices.device()));
  for (int i = 0; i < NDim; ++i) {
    outSpatialShape32.push_back(outSpatialShape[i]);
    kernelSize32.push_back(kernelSize[i]);
    if (subM) {
      stride32.push_back(1);
      padding32.push_back(kernelSize[i] / 2);
      dilation32.push_back(dilation[i]);
    } else {
      stride32.push_back(stride[i]);
      padding32.push_back(padding[i]);
      dilation32.push_back(dilation[i]);
    }
  }
  if (subM) {
    if (indices.device().type() == torch::kCPU) {
      auto getIndicePairFtor =
          functor::CreateSubMIndicePairFunctor<tv::CPU, int, int, NDim>();
      numActOut = getIndicePairFtor(
          tv::CPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(gridOut), tv::torch2tv<int>(indicePairs),
          tv::torch2tv<int>(indiceNum), kernelSize32, stride32, padding32,
          dilation32, outSpatialShape32, transpose);
      gridOut.fill_(-1);
    } else {
      auto getIndicePairFtor =
          functor::CreateSubMIndicePairFunctor<tv::TorchGPU, int, int, NDim>();
      numActOut = getIndicePairFtor(
          tv::TorchGPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(gridOut), tv::torch2tv<int>(indicePairs),
          tv::torch2tv<int>(indiceNum), kernelSize32, stride32, padding32,
          dilation32, outSpatialShape32, transpose, true);
    }
    return {indices, indicePairs, indiceNum};
  } else {
    torch::Tensor outInds =
        torch::zeros({numAct * kernelVolume, coorDim + 1},
                     torch::dtype(torch::kInt32).device(indices.device()));
    if (indices.device().type() == torch::kCPU) {
      auto getIndicePairFtor =
          functor::CreateConvIndicePairFunctor<tv::CPU, int, int, NDim>();
      numActOut = getIndicePairFtor(
          tv::CPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(outInds), tv::torch2tv<int>(gridOut),
          tv::torch2tv<int>(indicePairs), tv::torch2tv<int>(indiceNum),
          kernelSize32, stride32, padding32, dilation32, outSpatialShape32,
          transpose, true);
      gridOut.fill_(-1);
    } else {
      auto getIndicePairFtorP1 =
          functor::CreateConvIndicePairFunctorP1<tv::TorchGPU, int, int,
                                                 NDim>();
      auto getIndicePairFtorP2 =
          functor::CreateConvIndicePairFunctorP2<tv::TorchGPU, int, int,
                                                 NDim>();
      numActOut = getIndicePairFtorP1(
          tv::TorchGPU(), tv::torch2tv<const int>(indices),
          tv::torch2tv<int>(outInds), tv::torch2tv<int>(gridOut),
          tv::torch2tv<int>(indicePairs), tv::torch2tv<int>(indiceNum),
          tv::torch2tv<int>(indicePairUnique), kernelSize32, stride32,
          padding32, dilation32, outSpatialShape32, transpose);
      if (numActOut > 0) {
        auto res = torch::_unique(indicePairUnique);
        indicePairUnique = std::get<0>(res);
        numActOut = getIndicePairFtorP2(
            tv::TorchGPU(), tv::torch2tv<const int>(indices),
            tv::torch2tv<int>(outInds), tv::torch2tv<int>(gridOut),
            tv::torch2tv<int>(indicePairs), tv::torch2tv<int>(indiceNum),
            tv::torch2tv<int>(indicePairUnique), outSpatialShape32, transpose,
            true);
      }
    }
    return {outInds.slice(0, 0, numActOut), indicePairs, indiceNum};
  }
}

torch::Tensor IndiceConvForwardCUDAKernelLauncher(
    torch::Tensor features, torch::Tensor filters, torch::Tensor indicePairs,
    torch::Tensor indiceNum, int64_t numActOut, int64_t _inverse,
    int64_t _subM) {
  at::cuda::CUDAGuard device_guard(features.device());
  bool subM = _subM != 0;
  bool inverse = _inverse != 0;
  auto device = features.device().type();
  auto ndim = filters.dim() - 2;
  auto kernelVolume = indicePairs.size(0);
  auto numInPlanes = features.size(1);
  auto numOutPlanes = filters.size(ndim + 1);
  auto indicePairNumCpu = indiceNum.to({torch::kCPU});
  auto indicePairMaxSizeIter =
      std::max_element(indicePairNumCpu.data_ptr<int>(),
                       indicePairNumCpu.data_ptr<int>() + kernelVolume);
  int indicePairMaxOffset =
      indicePairMaxSizeIter - indicePairNumCpu.data_ptr<int>();
  int indicePairMaxSize = *indicePairMaxSizeIter;

  auto options =
      torch::TensorOptions().dtype(features.dtype()).device(features.device());

  torch::Tensor output = torch::zeros({numActOut, numOutPlanes}, options);
  torch::Tensor inputBuffer =
      torch::zeros({indicePairMaxSize, numInPlanes}, options);
  torch::Tensor outputBuffer =
      torch::zeros({indicePairMaxSize, numOutPlanes}, options);
  filters = filters.view({-1, numInPlanes, numOutPlanes});
  if (subM) {
    torch::mm_out(output, features, filters[indicePairMaxOffset]);
  }
  double totalGatherTime = 0;
  double totalGEMMTime = 0;
  double totalSAddTime = 0;
  for (int i = 0; i < kernelVolume; ++i) {
    auto nHot = indicePairNumCpu.data_ptr<int>()[i];
    if (nHot <= 0 || (subM && i == indicePairMaxOffset)) {
      continue;
    }

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        features.scalar_type(), "IndiceConvForwardKernel", [&] {
          auto outputBufferBlob = torch::from_blob(
              outputBuffer.data_ptr<scalar_t>(), {nHot, numOutPlanes}, options);
          auto inputBufferBlob = torch::from_blob(
              inputBuffer.data_ptr<scalar_t>(), {nHot, numInPlanes}, options);

          if (device == torch::kCPU) {
            functor::SparseGatherFunctor<tv::CPU, scalar_t, int> gatherFtor;
            gatherFtor(tv::CPU(), tv::torch2tv<scalar_t>(inputBuffer),
                       tv::torch2tv<const scalar_t>(features),
                       tv::torch2tv<const int>(indicePairs).subview(i, inverse),
                       nHot);
          } else {
            functor::SparseGatherFunctor<tv::TorchGPU, scalar_t, int>
                gatherFtor;
            gatherFtor(tv::TorchGPU(), tv::torch2tv<scalar_t>(inputBuffer),
                       tv::torch2tv<const scalar_t>(features),
                       tv::torch2tv<const int>(indicePairs).subview(i, inverse),
                       nHot);
            TV_CHECK_CUDA_ERR();
            /* slower than SparseGatherFunctor, may due to int->long conversion
            auto indicePairLong = indicePairs[i][inverse].to(torch::kInt64);
            auto indicePairBlob =
            torch::from_blob(indicePairLong.data_ptr<long>(), {nHot},
            indicePairOptions); torch::index_select_out(inputBufferBlob,
            features, 0, indicePairBlob);*/
          }
          torch::mm_out(outputBufferBlob, inputBufferBlob, filters[i]);

          if (device == torch::kCPU) {
            functor::SparseScatterAddFunctor<tv::CPU, scalar_t, int>
                scatterFtor;
            scatterFtor(
                tv::CPU(), tv::torch2tv<scalar_t>(output),
                tv::torch2tv<const scalar_t>(outputBuffer),
                tv::torch2tv<const int>(indicePairs).subview(i, !inverse), nHot,
                true);
          } else {
            functor::SparseScatterAddFunctor<tv::TorchGPU, scalar_t, int>
                scatterFtor;
            scatterFtor(
                tv::TorchGPU(), tv::torch2tv<scalar_t>(output),
                tv::torch2tv<const scalar_t>(outputBuffer),
                tv::torch2tv<const int>(indicePairs).subview(i, !inverse), nHot,
                true);
            TV_CHECK_CUDA_ERR();
          }
        });
  }
  return output;
}

std::vector<torch::Tensor> IndiceConvBackwardCUDAKernelLauncher(
    torch::Tensor features, torch::Tensor filters, torch::Tensor outGrad,
    torch::Tensor indicePairs, torch::Tensor indiceNum, int64_t _inverse,
    int64_t _subM) {
  at::cuda::CUDAGuard device_guard(features.device());
  bool subM = _subM != 0;
  bool inverse = _inverse != 0;

  auto device = features.device().type();
  auto ndim = filters.dim() - 2;
  auto kernelVolume = indicePairs.size(0);
  auto numInPlanes = features.size(1);
  auto numOutPlanes = filters.size(ndim + 1);
  auto indicePairNumCpu = indiceNum.to({torch::kCPU});
  auto indicePairMaxSizeIter =
      std::max_element(indicePairNumCpu.data_ptr<int>(),
                       indicePairNumCpu.data_ptr<int>() + kernelVolume);
  int indicePairMaxOffset =
      indicePairMaxSizeIter - indicePairNumCpu.data_ptr<int>();
  int indicePairMaxSize = *indicePairMaxSizeIter;
  auto options =
      torch::TensorOptions().dtype(features.dtype()).device(features.device());
  auto filterShape = filters.sizes();
  torch::Tensor inputGrad = torch::zeros(features.sizes(), options);
  torch::Tensor filtersGrad = torch::zeros(filterShape, options);
  torch::Tensor inputBuffer =
      torch::zeros({indicePairMaxSize, numInPlanes}, options);
  torch::Tensor outputBuffer =
      torch::zeros({indicePairMaxSize, numOutPlanes}, options);

  filters = filters.view({-1, numInPlanes, numOutPlanes});
  filtersGrad = filtersGrad.view({-1, numInPlanes, numOutPlanes});
  if (subM) {
    auto filterGradSub = filtersGrad[indicePairMaxOffset];
    torch::mm_out(filterGradSub, features.t(), outGrad);
    torch::mm_out(inputGrad, outGrad, filters[indicePairMaxOffset].t());
  }
  for (int i = 0; i < kernelVolume; ++i) {
    auto nHot = indicePairNumCpu.data_ptr<int>()[i];
    if (nHot <= 0 || (subM && i == indicePairMaxOffset)) {
      continue;
    }

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        features.scalar_type(), "IndiceConvBackwardKernel", [&] {
          if (device == torch::kCPU) {
            functor::SparseGatherFunctor<tv::CPU, scalar_t, int> gatherFtor;
            functor::SparseGatherFunctor<tv::CPU, scalar_t, int> gatherFtorOut;
            gatherFtor(tv::CPU(), tv::torch2tv<scalar_t>(inputBuffer),
                       tv::torch2tv<const scalar_t>(features),
                       tv::torch2tv<const int>(indicePairs).subview(i, inverse),
                       nHot);
            gatherFtorOut(
                tv::CPU(), tv::torch2tv<scalar_t>(outputBuffer),
                tv::torch2tv<const scalar_t>(outGrad),
                tv::torch2tv<const int>(indicePairs).subview(i, !inverse),
                nHot);
          } else {
            functor::SparseGatherFunctor<tv::TorchGPU, scalar_t, int>
                gatherFtor;
            functor::SparseGatherFunctor<tv::TorchGPU, scalar_t, int>
                gatherFtorOut;
            gatherFtor(tv::TorchGPU(), tv::torch2tv<scalar_t>(inputBuffer),
                       tv::torch2tv<const scalar_t>(features),
                       tv::torch2tv<const int>(indicePairs).subview(i, inverse),
                       nHot);
            TV_CHECK_CUDA_ERR();
            gatherFtorOut(
                tv::TorchGPU(), tv::torch2tv<scalar_t>(outputBuffer),
                tv::torch2tv<const scalar_t>(outGrad),
                tv::torch2tv<const int>(indicePairs).subview(i, !inverse),
                nHot);
            TV_CHECK_CUDA_ERR();
          }
          auto filterGradSub = filtersGrad[i];
          auto outputBufferBlob = torch::from_blob(
              outputBuffer.data_ptr<scalar_t>(), {nHot, numOutPlanes}, options);
          auto inputBufferBlob = torch::from_blob(
              inputBuffer.data_ptr<scalar_t>(), {nHot, numInPlanes}, options);

          torch::mm_out(filterGradSub, inputBufferBlob.t(), outputBufferBlob);
          torch::mm_out(inputBufferBlob, outputBufferBlob, filters[i].t());
          if (device == torch::kCPU) {
            functor::SparseScatterAddFunctor<tv::CPU, scalar_t, int>
                scatterFtor;
            scatterFtor(
                tv::CPU(), tv::torch2tv<scalar_t>(inputGrad),
                tv::torch2tv<const scalar_t>(inputBuffer),
                tv::torch2tv<const int>(indicePairs).subview(i, inverse), nHot);
          } else {
            functor::SparseScatterAddFunctor<tv::TorchGPU, scalar_t, int>
                scatterFtor;
            scatterFtor(
                tv::TorchGPU(), tv::torch2tv<scalar_t>(inputGrad),
                tv::torch2tv<const scalar_t>(inputBuffer),
                tv::torch2tv<const int>(indicePairs).subview(i, inverse), nHot);
            TV_CHECK_CUDA_ERR();
          }
        });
  }
  return {inputGrad, filtersGrad.view(filterShape)};
}

template std::vector<torch::Tensor> GetIndicePairsForwardCUDAKernelLauncher<2>(
    torch::Tensor indices, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose);

template std::vector<torch::Tensor> GetIndicePairsForwardCUDAKernelLauncher<3>(
    torch::Tensor indices, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose);

template std::vector<torch::Tensor> GetIndicePairsForwardCUDAKernelLauncher<4>(
    torch::Tensor indices, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose);

template std::vector<torch::Tensor> GetIndicePairsBackwardCUDAKernelLauncher<2>(
    torch::Tensor indices, torch::Tensor gridOut, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose);

template std::vector<torch::Tensor> GetIndicePairsBackwardCUDAKernelLauncher<3>(
    torch::Tensor indices, torch::Tensor gridOut, int64_t batchSize,
    std::vector<int64_t> outSpatialShape, std::vector<int64_t> spatialShape,
    std::vector<int64_t> kernelSize, std::vector<int64_t> stride,
    std::vector<int64_t> padding, std::vector<int64_t> dilation,
    std::vector<int64_t> outPadding, int64_t _subM, int64_t _transpose);
