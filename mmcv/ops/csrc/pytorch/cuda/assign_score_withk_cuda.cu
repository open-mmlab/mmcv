// Modified from
// https://github.com/CVMI-Lab/PAConv/tree/main/scene_seg/lib/paconv_lib/src/gpu
#include <stdio.h>
#include <stdlib.h>

#include "assign_score_withk_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void AssignScoreWithKForwardCUDAKernelLauncher(
    int B, int N0, int N1, int M, int K, int O, int aggregate,
    const Tensor& points, const Tensor& centers, const Tensor& scores,
    const Tensor& knn_idx, Tensor& output) {
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks(DIVUP(B * O * N1 * K, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      points.scalar_type(), "assign_score_withk_forward_cuda_kernel", [&] {
        assign_score_withk_forward_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                B, N0, N1, M, K, O, aggregate, points.data_ptr<scalar_t>(),
                centers.data_ptr<scalar_t>(), scores.data_ptr<scalar_t>(),
                knn_idx.data_ptr<int64_t>(), output.data_ptr<scalar_t>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void AssignScoreWithKBackwardCUDAKernelLauncher(
    int B, int N0, int N1, int M, int K, int O, int aggregate,
    const Tensor& grad_out, const Tensor& points, const Tensor& centers,
    const Tensor& scores, const Tensor& knn_idx, Tensor& grad_points,
    Tensor& grad_centers, Tensor& grad_scores) {
  at::cuda::CUDAGuard device_guard(grad_out.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks1(DIVUP(B * M * O, THREADS_PER_BLOCK));
  dim3 threads1(THREADS_PER_BLOCK);
  dim3 blocks2(DIVUP(B * N1 * K * M, THREADS_PER_BLOCK));
  dim3 threads2(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_out.scalar_type(), "assign_score_withk_points_backward_cuda_kernel",
      [&] {
        assign_score_withk_points_backward_cuda_kernel<scalar_t>
            <<<blocks1, threads1, 0, stream>>>(
                B, N0, N1, M, K, O, aggregate, grad_out.data_ptr<scalar_t>(),
                scores.data_ptr<scalar_t>(), knn_idx.data_ptr<int64_t>(),
                grad_points.data_ptr<scalar_t>(),
                grad_centers.data_ptr<scalar_t>());
      });

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_out.scalar_type(), "assign_score_withk_scores_backward_cuda_kernel",
      [&] {
        assign_score_withk_scores_backward_cuda_kernel<scalar_t>
            <<<blocks2, threads2, 0, stream>>>(
                B, N0, N1, M, K, O, aggregate, grad_out.data_ptr<scalar_t>(),
                points.data_ptr<scalar_t>(), centers.data_ptr<scalar_t>(),
                knn_idx.data_ptr<int64_t>(), grad_scores.data_ptr<scalar_t>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}
