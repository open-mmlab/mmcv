#include "hip/hip_runtime.h"
// Copyright 2019 Yan Yan
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <ATen/ATen.h>
#include <utils/spconv/spconv/mp_helper.h>
#include <utils/spconv/spconv/reordering.h>
#include <utils/spconv/tensorview/helper_launch.h>
#include <utils/spconv/tensorview/tensorview.h>

#include <chrono>
#include <limits>
#include <spconv/reordering.cuh>
#include <type_traits>
#include <utils/spconv/tensorview/helper_kernel.cuh>

#include "../spconv_utils.h"
#include "pytorch_cuda_helper.hpp"

namespace functor {
template <typename scalar_t, typename Index>
struct SparseGatherFunctor<tv::TorchGPU, scalar_t, Index> {
  using vecload_type_t =
      std::conditional_t<std::is_same<scalar_t, at::Half>::value, int2, int4>;
  using kernel_block_t = mp_list_c<int, 64, 32, 16>;
  void operator()(const tv::TorchGPU &d, tv::TensorView<scalar_t> buffer,
                  tv::TensorView<const scalar_t> features,
                  tv::TensorView<const Index> indices, int size) {
    if (size <= 0) return;
    int numPlanes = features.dim(1);
    bool notFound = true;
    constexpr int vecloadFactor = sizeof(vecload_type_t) / sizeof(scalar_t);
    mp_for_each<kernel_block_t>([=, &buffer, &features, &indices,
                                 &notFound](auto NumTLP) {
      constexpr int NumILP = NumTLP / 4;
      int nHotBlock = (size / NumTLP) * NumTLP;
      if (notFound) {
        if (numPlanes % NumTLP == 0) {
          if (nHotBlock >= NumTLP) {
            gatherVecBlockKernel<scalar_t, Index, int(NumTLP), NumILP,
                                 vecload_type_t>
                <<<dim3(numPlanes / NumTLP, size / NumTLP),
                   dim3(NumTLP / vecloadFactor, NumTLP / NumILP), 0,
                   d.getStream()>>>(buffer.data(), features.data(),
                                    indices.data(), nHotBlock,
                                    numPlanes / vecloadFactor);

            TV_CHECK_CUDA_ERR();
          }
          if (size - nHotBlock > 0) {
            gatherVecKernel<scalar_t, Index, int(NumTLP), NumILP,
                            vecload_type_t>
                <<<dim3(1, numPlanes / NumTLP),
                   dim3(NumTLP / NumILP, NumTLP / vecloadFactor), 0,
                   d.getStream()>>>(buffer.data() + nHotBlock * numPlanes,
                                    features.data(), indices.data() + nHotBlock,
                                    size - nHotBlock,
                                    numPlanes / vecloadFactor);
            TV_CHECK_CUDA_ERR();
          }
          notFound = false;
        }
      }
    });

    if (notFound) {
      constexpr int NumTLP = 64;
      constexpr int NumILP = NumTLP / 4;
      gatherGenericKernel<scalar_t, Index, NumTLP, NumILP>
          <<<dim3(tv::launch::DivUp(size, NumTLP),
                  tv::launch::DivUp(numPlanes, NumTLP)),
             dim3(NumTLP / NumILP, NumTLP), 0, d.getStream()>>>(
              buffer.data(), features.data(), indices.data(), size, numPlanes);
      TV_CHECK_CUDA_ERR();
    }
  }
};
template <typename scalar_t, typename Index>
struct SparseScatterAddFunctor<tv::TorchGPU, scalar_t, Index> {
  using vecload_type_t =
      std::conditional_t<std::is_same<scalar_t, at::Half>::value, int2, int4>;
  using kernel_block_t = mp_list_c<int, 64, 32, 16>;
  void operator()(const tv::TorchGPU &d, tv::TensorView<scalar_t> outFeatures,
                  tv::TensorView<const scalar_t> buffer,
                  tv::TensorView<const Index> indices, int size, bool stable) {
    if (size <= 0) return;
    int numPlanes = outFeatures.dim(1);
    bool notFound = true;
    constexpr int vecloadFactor =
        sizeof(vecload_type_t) / sizeof(scalar_t);  // important for half.
    mp_for_each<kernel_block_t>([=, &d, &outFeatures, &buffer, &indices,
                                 &notFound](auto NumTLP) {
      constexpr int NumILP = NumTLP / 4;
      int nHotBlock = (size / NumTLP) * NumTLP;
      if (notFound) {
        if (numPlanes % NumTLP == 0) {
          if (nHotBlock >= NumTLP) {
            scatterAddVecBlockKernel<scalar_t, Index, int(NumTLP), NumILP,
                                     vecload_type_t>
                <<<dim3(numPlanes / NumTLP, size / NumTLP),
                   dim3(NumTLP / vecloadFactor, NumTLP / NumILP), 0,
                   d.getStream()>>>(outFeatures.data(), buffer.data(),
                                    indices.data(), nHotBlock,
                                    numPlanes / vecloadFactor);
            TV_CHECK_CUDA_ERR();
          }
          if (size - nHotBlock > 0) {
            scatterAddGenericKernel<scalar_t, Index, int(NumTLP), NumILP>
                <<<dim3(1, numPlanes / NumTLP), dim3(NumTLP / NumILP, NumTLP),
                   0, d.getStream()>>>(
                    outFeatures.data(), buffer.data() + nHotBlock * numPlanes,
                    indices.data() + nHotBlock, size - nHotBlock, numPlanes);
            TV_CHECK_CUDA_ERR();
          }
          notFound = false;
        }
      }
    });
    if (notFound) {
      constexpr int NumTLP = 64;
      constexpr int NumILP = NumTLP / 4;
      scatterAddGenericKernel<scalar_t, Index, NumTLP, NumILP>
          <<<dim3(tv::launch::DivUp(size, NumTLP),
                  tv::launch::DivUp(numPlanes, NumTLP)),
             dim3(NumTLP / NumILP, NumTLP), 0, d.getStream()>>>(
              outFeatures.data(), buffer.data(), indices.data(), size,
              numPlanes);
      TV_CHECK_CUDA_ERR();
    }
  }
};

}  // namespace functor

#define DECLARE_GPU_SPECS_T_INDEX(scalar_t, Index)                             \
  template struct functor::SparseGatherFunctor<tv::TorchGPU, scalar_t, Index>; \
  template struct functor::SparseScatterAddFunctor<tv::TorchGPU, scalar_t,     \
                                                   Index>;

#define DECLARE_GPU_SPECS(scalar_t) DECLARE_GPU_SPECS_T_INDEX(scalar_t, int);

DECLARE_GPU_SPECS(float);
DECLARE_GPU_SPECS(double);
DECLARE_GPU_SPECS(at::Half);

#undef DECLARE_GPU_SPECS
#undef DECLARE_GPU_SPECS_T_INDEX
