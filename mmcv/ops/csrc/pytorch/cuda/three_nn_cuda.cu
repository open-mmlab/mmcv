// Modified from
// https://github.com/sshaoshuai/Pointnet2.PyTorch/tree/master/pointnet2/src/interpolate_gpu.cu

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "pytorch_cuda_helper.hpp"
#include "three_nn_cuda_kernel.cuh"

void ThreeNNForwardCUDAKernelLauncher(int b, int n, int m, const Tensor unknown,
                                      const Tensor known, Tensor dist2,
                                      Tensor idx) {
  // unknown: (B, N, 3)
  // known: (B, M, 3)
  // output:
  //      dist2: (B, N, 3)
  //      idx: (B, N, 3)

  at::cuda::CUDAGuard device_guard(unknown.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), b);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      unknown.scalar_type(), "three_nn_forward_cuda_kernel", [&] {
        three_nn_forward_cuda_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            b, n, m, unknown.data_ptr<scalar_t>(), known.data_ptr<scalar_t>(),
            dist2.data_ptr<scalar_t>(), idx.data_ptr<int>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}
