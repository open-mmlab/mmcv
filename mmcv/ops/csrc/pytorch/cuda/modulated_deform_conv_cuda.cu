// Copyright (c) OpenMMLab. All rights reserved
#include "modulated_deform_conv_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void modulated_deformable_im2col_cuda(
    const Tensor data_im, const Tensor data_offset, const Tensor data_mask,
    const int batch_size, const int channels, const int height_im,
    const int width_im, const int height_col, const int width_col,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int deformable_group, Tensor data_col) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels = channels * batch_size * height_col * width_col;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "modulated_deformable_im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t *data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t *data_col_ = data_col.data_ptr<scalar_t>();

        modulated_deformable_im2col_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_im_, data_offset_, data_mask_, height_im,
            width_im, kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w,
            dilation_h, dilation_w, channel_per_deformable_group, batch_size,
            channels, deformable_group, height_col, width_col, data_col_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void modulated_deformable_col2im_cuda(
    const Tensor data_col, const Tensor data_offset, const Tensor data_mask,
    const int batch_size, const int channels, const int height_im,
    const int width_im, const int height_col, const int width_col,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int deformable_group, Tensor grad_im) {
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels =
      channels * kernel_h * kernel_w * batch_size * height_col * width_col;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "modulated_deformable_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t *data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        modulated_deformable_col2im_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, data_offset_, data_mask_, channels,
            height_im, width_im, kernel_h, kernel_w, pad_h, pad_w, stride_h,
            stride_w, dilation_h, dilation_w, channel_per_deformable_group,
            batch_size, deformable_group, height_col, width_col, grad_im_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void modulated_deformable_col2im_coord_cuda(
    const Tensor data_col, const Tensor data_im, const Tensor data_offset,
    const Tensor data_mask, const int batch_size, const int channels,
    const int height_im, const int width_im, const int height_col,
    const int width_col, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int deformable_group,
    Tensor grad_offset, Tensor grad_mask) {
  const int num_kernels = batch_size * height_col * width_col * 2 * kernel_h *
                          kernel_w * deformable_group;
  const int channel_per_deformable_group =
      channels * kernel_h * kernel_w / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "modulated_deformable_col2im_coord_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t *data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t *grad_offset_ = grad_offset.data_ptr<scalar_t>();
        scalar_t *grad_mask_ = grad_mask.data_ptr<scalar_t>();

        modulated_deformable_col2im_coord_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, data_im_, data_offset_, data_mask_,
            channels, height_im, width_im, kernel_h, kernel_w, pad_h, pad_w,
            stride_h, stride_w, dilation_h, dilation_w,
            channel_per_deformable_group, batch_size,
            2 * kernel_h * kernel_w * deformable_group, deformable_group,
            height_col, width_col, grad_offset_, grad_mask_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
