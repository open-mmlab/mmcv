#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved
#include "pytorch_cuda_helper.hpp"
#include "pytorch_device_registry.hpp"
#include "sync_bn_cuda_kernel.cuh"

void SyncBNForwardMeanCUDAKernelLauncher(const Tensor input, Tensor mean) {
  int num = input.size(0);
  int channels = input.size(1);
  int spatial = input.size(2);

  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "sync_bn_forward_mean_cuda_kernel", [&] {
        sync_bn_forward_mean_cuda_kernel<scalar_t>
            <<<channels, THREADS_PER_BLOCK, 0, stream>>>(
                input.data_ptr<scalar_t>(), mean.data_ptr<float>(), num,
                channels, spatial);
      });
  AT_CUDA_CHECK(hipGetLastError());
}

void SyncBNForwardVarCUDAKernelLauncher(const Tensor input, const Tensor mean,
                                        Tensor var) {
  int num = input.size(0);
  int channels = input.size(1);
  int spatial = input.size(2);

  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "sync_bn_forward_mean_cuda_kernel", [&] {
        sync_bn_forward_var_cuda_kernel<scalar_t>
            <<<channels, THREADS_PER_BLOCK, 0, stream>>>(
                input.data_ptr<scalar_t>(), mean.data_ptr<float>(),
                var.data_ptr<float>(), num, channels, spatial);
      });
  AT_CUDA_CHECK(hipGetLastError());
}

void SyncBNForwardOutputCUDAKernelLauncher(
    const Tensor input, const Tensor mean, const Tensor var,
    Tensor running_mean, Tensor running_var, const Tensor weight,
    const Tensor bias, Tensor norm, Tensor std, Tensor output, float eps,
    float momentum, int group_size) {
  int num = input.size(0);
  int channels = input.size(1);
  int spatial = input.size(2);

  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "sync_bn_forward_mean_cuda_kernel", [&] {
        sync_bn_forward_output_cuda_kernel<scalar_t>
            <<<channels, THREADS_PER_BLOCK, 0, stream>>>(
                input.data_ptr<scalar_t>(), mean.data_ptr<float>(),
                var.data_ptr<float>(), running_mean.data_ptr<float>(),
                running_var.data_ptr<float>(), weight.data_ptr<float>(),
                bias.data_ptr<float>(), norm.data_ptr<float>(),
                std.data_ptr<float>(), output.data_ptr<scalar_t>(), num,
                channels, spatial, eps, momentum, group_size);
      });
  AT_CUDA_CHECK(hipGetLastError());
}

void SyncBNBackwardParamCUDAKernelLauncher(const Tensor grad_output,
                                           const Tensor norm,
                                           Tensor grad_weight,
                                           Tensor grad_bias) {
  int num = grad_output.size(0);
  int channels = grad_output.size(1);
  int spatial = grad_output.size(2);

  at::cuda::CUDAGuard device_guard(grad_output.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "sync_bn_backward_param_cuda_kernel", [&] {
        sync_bn_backward_param_cuda_kernel<scalar_t>
            <<<channels, THREADS_PER_BLOCK, 0, stream>>>(
                grad_output.data_ptr<scalar_t>(), norm.data_ptr<float>(),
                grad_weight.data_ptr<float>(), grad_bias.data_ptr<float>(), num,
                channels, spatial);
      });
  AT_CUDA_CHECK(hipGetLastError());
}

void SyncBNBackwardDataCUDAKernelLauncher(const Tensor grad_output,
                                          const Tensor weight,
                                          const Tensor grad_weight,
                                          const Tensor grad_bias,
                                          const Tensor norm, const Tensor std,
                                          Tensor grad_input) {
  int output_size = grad_input.numel();
  int num = grad_input.size(0);
  int channels = grad_input.size(1);
  int spatial = grad_input.size(2);

  at::cuda::CUDAGuard device_guard(grad_input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "sync_bn_backward_data_cuda_kernel", [&] {
        sync_bn_backward_data_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, grad_output.data_ptr<scalar_t>(),
                weight.data_ptr<float>(), grad_weight.data_ptr<float>(),
                grad_bias.data_ptr<float>(), norm.data_ptr<float>(),
                std.data_ptr<float>(), grad_input.data_ptr<scalar_t>(), num,
                channels, spatial);
      });
  AT_CUDA_CHECK(hipGetLastError());
}

void sync_bn_forward_mean_cuda(const Tensor input, Tensor mean) {
  SyncBNForwardMeanCUDAKernelLauncher(input, mean);
}

void sync_bn_forward_var_cuda(const Tensor input, const Tensor mean,
                              Tensor var) {
  SyncBNForwardVarCUDAKernelLauncher(input, mean, var);
}

void sync_bn_forward_output_cuda(const Tensor input, const Tensor mean,
                                 const Tensor var, Tensor running_mean,
                                 Tensor running_var, const Tensor weight,
                                 const Tensor bias, Tensor norm, Tensor std,
                                 Tensor output, float eps, float momentum,
                                 int group_size) {
  SyncBNForwardOutputCUDAKernelLauncher(input, mean, var, running_mean,
                                        running_var, weight, bias, norm, std,
                                        output, eps, momentum, group_size);
}

void sync_bn_backward_param_cuda(const Tensor grad_output, const Tensor norm,
                                 Tensor grad_weight, Tensor grad_bias) {
  SyncBNBackwardParamCUDAKernelLauncher(grad_output, norm, grad_weight,
                                        grad_bias);
}

void sync_bn_backward_data_cuda(const Tensor grad_output, const Tensor weight,
                                const Tensor grad_weight,
                                const Tensor grad_bias, const Tensor norm,
                                const Tensor std, Tensor grad_input) {
  SyncBNBackwardDataCUDAKernelLauncher(grad_output, weight, grad_weight,
                                       grad_bias, norm, std, grad_input);
}

void sync_bn_forward_mean_impl(const Tensor input, Tensor mean);

void sync_bn_forward_var_impl(const Tensor input, const Tensor mean,
                              Tensor var);

void sync_bn_forward_output_impl(const Tensor input, const Tensor mean,
                                 const Tensor var, Tensor running_mean,
                                 Tensor running_var, const Tensor weight,
                                 const Tensor bias, Tensor norm, Tensor std,
                                 Tensor output, float eps, float momentum,
                                 int group_size);

void sync_bn_backward_param_impl(const Tensor grad_output, const Tensor norm,
                                 Tensor grad_weight, Tensor grad_bias);

void sync_bn_backward_data_impl(const Tensor grad_output, const Tensor weight,
                                const Tensor grad_weight,
                                const Tensor grad_bias, const Tensor norm,
                                const Tensor std, Tensor grad_input);

REGISTER_DEVICE_IMPL(sync_bn_forward_mean_impl, CUDA,
                     sync_bn_forward_mean_cuda);
REGISTER_DEVICE_IMPL(sync_bn_forward_var_impl, CUDA, sync_bn_forward_var_cuda);
REGISTER_DEVICE_IMPL(sync_bn_forward_output_impl, CUDA,
                     sync_bn_forward_output_cuda);
REGISTER_DEVICE_IMPL(sync_bn_backward_param_impl, CUDA,
                     sync_bn_backward_param_cuda);
REGISTER_DEVICE_IMPL(sync_bn_backward_data_impl, CUDA,
                     sync_bn_backward_data_cuda);
