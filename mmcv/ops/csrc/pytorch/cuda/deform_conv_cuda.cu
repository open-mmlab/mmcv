// Copyright (c) OpenMMLab. All rights reserved
#include "deform_conv_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void deformable_im2col_cuda(Tensor data_im, Tensor data_offset,
                            const int channels, const int height,
                            const int width, const int ksize_h,
                            const int ksize_w, const int pad_h, const int pad_w,
                            const int stride_h, const int stride_w,
                            const int dilation_h, const int dilation_w,
                            const int parallel_imgs, const int deformable_group,
                            Tensor data_col) {
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col * parallel_imgs;
  int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "deformable_im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t *data_col_ = data_col.data_ptr<scalar_t>();

        deformable_im2col_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_im_, data_offset_, height, width, ksize_h,
            ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
            channel_per_deformable_group, parallel_imgs, channels,
            deformable_group, height_col, width_col, data_col_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void deformable_col2im_cuda(Tensor data_col, Tensor data_offset,
                            const int channels, const int height,
                            const int width, const int ksize_h,
                            const int ksize_w, const int pad_h, const int pad_w,
                            const int stride_h, const int stride_w,
                            const int dilation_h, const int dilation_w,
                            const int parallel_imgs, const int deformable_group,
                            Tensor grad_im) {
  // todo: make sure parallel_imgs is passed in correctly
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels =
      channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;
  int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        deformable_col2im_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, data_offset_, channels, height, width,
            ksize_h, ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h,
            dilation_w, channel_per_deformable_group, parallel_imgs,
            deformable_group, height_col, width_col, grad_im_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void deformable_col2im_coord_cuda(
    Tensor data_col, Tensor data_im, Tensor data_offset, const int channels,
    const int height, const int width, const int ksize_h, const int ksize_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int parallel_imgs,
    const int deformable_group, Tensor grad_offset) {
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = height_col * width_col * 2 * ksize_h * ksize_w *
                    deformable_group * parallel_imgs;
  int channel_per_deformable_group =
      channels * ksize_h * ksize_w / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_coord_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t *grad_offset_ = grad_offset.data_ptr<scalar_t>();

        deformable_col2im_coord_gpu_kernel<<<
            GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0,
            at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, data_im_, data_offset_, channels, height,
            width, ksize_h, ksize_w, pad_h, pad_w, stride_h, stride_w,
            dilation_h, dilation_w, channel_per_deformable_group, parallel_imgs,
            2 * ksize_h * ksize_w * deformable_group, deformable_group,
            height_col, width_col, grad_offset_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}
