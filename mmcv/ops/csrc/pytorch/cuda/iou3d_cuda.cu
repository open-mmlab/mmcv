#include "hip/hip_runtime.h"
// Modified from
// https://github.com/open-mmlab/OpenPCDet/blob/master/pcdet/ops/iou3d_nms/src/iou3d_nms_kernel.cu

/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/

#include <stdio.h>

#include "iou3d_cuda_kernel.cuh"
#include "nms_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void IoU3DBoxesOverlapBevForwardCUDAKernelLauncher(const int num_a,
                                                   const Tensor boxes_a,
                                                   const int num_b,
                                                   const Tensor boxes_b,
                                                   Tensor ans_overlap) {
  at::cuda::CUDAGuard device_guard(boxes_a.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(GET_BLOCKS(num_b, THREADS_PER_BLOCK_IOU3D),
              GET_BLOCKS(num_a, THREADS_PER_BLOCK_IOU3D));
  dim3 threads(THREADS_PER_BLOCK_IOU3D, THREADS_PER_BLOCK_IOU3D);

  iou3d_boxes_overlap_bev_forward_cuda_kernel<<<blocks, threads, 0, stream>>>(
      num_a, boxes_a.data_ptr<float>(), num_b, boxes_b.data_ptr<float>(),
      ans_overlap.data_ptr<float>());

  AT_CUDA_CHECK(hipGetLastError());
}

void IoU3DNMS3DForwardCUDAKernelLauncher(const Tensor boxes, Tensor& keep,
                                         Tensor& keep_num,
                                         float nms_overlap_thresh) {
  using namespace at::indexing;
  at::cuda::CUDAGuard device_guard(boxes.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int boxes_num = boxes.size(0);

  const int col_blocks =
      (boxes_num + THREADS_PER_BLOCK_NMS - 1) / THREADS_PER_BLOCK_NMS;
  Tensor mask =
      at::empty({boxes_num, col_blocks}, boxes.options().dtype(at::kLong));

  dim3 blocks(GET_BLOCKS(boxes_num, THREADS_PER_BLOCK_NMS),
              GET_BLOCKS(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);

  iou3d_nms3d_forward_cuda_kernel<<<blocks, threads, 0, stream>>>(
      boxes_num, nms_overlap_thresh, boxes.data_ptr<float>(),
      (unsigned long long*)mask.data_ptr<int64_t>());

  at::Tensor keep_t = at::zeros(
      {boxes_num}, boxes.options().dtype(at::kBool).device(at::kCUDA));
  gather_keep_from_mask<<<1, min(col_blocks, THREADS_PER_BLOCK),
                          col_blocks * sizeof(unsigned long long), stream>>>(
      keep_t.data_ptr<bool>(), (unsigned long long*)mask.data_ptr<int64_t>(),
      boxes_num);

  auto keep_data = keep_t.nonzero().index({Slice(), 0});
  keep_num.fill_(at::Scalar(keep_data.size(0)));
  keep.index_put_({Slice(0, keep_data.size(0))}, keep_data);
  AT_CUDA_CHECK(hipGetLastError());
}

void IoU3DNMS3DNormalForwardCUDAKernelLauncher(const Tensor boxes, Tensor& keep,
                                               Tensor& keep_num,
                                               float nms_overlap_thresh) {
  using namespace at::indexing;
  at::cuda::CUDAGuard device_guard(boxes.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int boxes_num = boxes.size(0);

  const int col_blocks =
      (boxes_num + THREADS_PER_BLOCK_NMS - 1) / THREADS_PER_BLOCK_NMS;
  Tensor mask =
      at::empty({boxes_num, col_blocks}, boxes.options().dtype(at::kLong));

  dim3 blocks(GET_BLOCKS(boxes_num, THREADS_PER_BLOCK_NMS),
              GET_BLOCKS(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);

  iou3d_nms3d_normal_forward_cuda_kernel<<<blocks, threads, 0, stream>>>(
      boxes_num, nms_overlap_thresh, boxes.data_ptr<float>(),
      (unsigned long long*)mask.data_ptr<int64_t>());

  at::Tensor keep_t = at::zeros(
      {boxes_num}, boxes.options().dtype(at::kBool).device(at::kCUDA));
  gather_keep_from_mask<<<1, min(col_blocks, THREADS_PER_BLOCK),
                          col_blocks * sizeof(unsigned long long), stream>>>(
      keep_t.data_ptr<bool>(), (unsigned long long*)mask.data_ptr<int64_t>(),
      boxes_num);

  auto keep_data = keep_t.nonzero().index({Slice(), 0});
  keep_num.fill_(at::Scalar(keep_data.size(0)));
  keep.index_put_({Slice(0, keep_data.size(0))}, keep_data);
  AT_CUDA_CHECK(hipGetLastError());
}
