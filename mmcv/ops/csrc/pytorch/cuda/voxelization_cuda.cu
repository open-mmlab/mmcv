#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.
#include <stdio.h>
#include <stdlib.h>

#include "pytorch_cuda_helper.hpp"
#include "voxelization_cuda_kernel.cuh"

int HardVoxelizeForwardCUDAKernelLauncher(
    const at::Tensor &points, at::Tensor &voxels, at::Tensor &coors,
    at::Tensor &num_points_per_voxel, const std::vector<float> voxel_size,
    const std::vector<float> coors_range, const int max_points,
    const int max_voxels, const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  // map points to voxel coors
  at::Tensor temp_coors =
      at::zeros({num_points, NDim}, points.options().dtype(at::kInt));

  dim3 grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 block(512);

  // 1. link point to corresponding voxel coors
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "hard_voxelize_kernel", ([&] {
        dynamic_voxelize_kernel<scalar_t, int><<<grid, block, 0, stream>>>(
            points.contiguous().data_ptr<scalar_t>(),
            temp_coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
            coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
            coors_z_max, grid_x, grid_y, grid_z, num_points, num_features,
            NDim);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  // 2. map point to the idx of the corresponding voxel, find duplicate coor
  // create some temporary variables
  auto point_to_pointidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto point_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));

  dim3 map_grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 map_block(512);

  AT_DISPATCH_ALL_TYPES(
      temp_coors.scalar_type(), "determin_duplicate", ([&] {
        point_to_voxelidx_kernel<int><<<map_grid, map_block, 0, stream>>>(
            temp_coors.contiguous().data_ptr<int>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            point_to_pointidx.contiguous().data_ptr<int>(), max_points,
            max_voxels, num_points, NDim);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  // 3. determine voxel num and voxel's coor index
  // make the logic in the CUDA device could accelerate about 10 times
  auto coor_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto voxel_num = at::zeros(
      {
          1,
      },
      points.options().dtype(at::kInt));  // must be zero from the beginning

  AT_DISPATCH_ALL_TYPES(temp_coors.scalar_type(), "determin_duplicate", ([&] {
                          determin_voxel_num<int><<<1, 1, 0, stream>>>(
                              num_points_per_voxel.contiguous().data_ptr<int>(),
                              point_to_voxelidx.contiguous().data_ptr<int>(),
                              point_to_pointidx.contiguous().data_ptr<int>(),
                              coor_to_voxelidx.contiguous().data_ptr<int>(),
                              voxel_num.contiguous().data_ptr<int>(),
                              max_points, max_voxels, num_points);
                        }));

  AT_CUDA_CHECK(hipGetLastError());

  // 4. copy point features to voxels
  // Step 4 & 5 could be parallel
  auto pts_output_size = num_points * num_features;
  dim3 cp_grid(std::min(at::cuda::ATenCeilDiv(pts_output_size, 512), 4096));
  dim3 cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_point_to_voxel<float, int><<<cp_grid, cp_block, 0, stream>>>(
            pts_output_size, points.contiguous().data_ptr<float>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            coor_to_voxelidx.contiguous().data_ptr<int>(),
            voxels.contiguous().data_ptr<float>(), max_points, num_features,
            num_points, NDim);
      }));
  //   hipDeviceSynchronize();
  //   AT_CUDA_CHECK(hipGetLastError());

  // 5. copy coors of each voxels
  auto coors_output_size = num_points * NDim;
  dim3 coors_cp_grid(
      std::min(at::cuda::ATenCeilDiv(coors_output_size, 512), 4096));
  dim3 coors_cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_voxel_coors<float, int>
            <<<coors_cp_grid, coors_cp_block, 0, stream>>>(
                coors_output_size, temp_coors.contiguous().data_ptr<int>(),
                point_to_voxelidx.contiguous().data_ptr<int>(),
                coor_to_voxelidx.contiguous().data_ptr<int>(),
                coors.contiguous().data_ptr<int>(), num_points, NDim);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  auto voxel_num_cpu = voxel_num.to(at::kCPU);
  int voxel_num_int = voxel_num_cpu.data_ptr<int>()[0];

  return voxel_num_int;
}

void DynamicVoxelizeForwardCUDAKernelLauncher(
    const at::Tensor &points, at::Tensor &coors,
    const std::vector<float> voxel_size, const std::vector<float> coors_range,
    const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  const int col_blocks = at::cuda::ATenCeilDiv(num_points, THREADS_PER_BLOCK);
  dim3 blocks(col_blocks);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_ALL_TYPES(points.scalar_type(), "dynamic_voxelize_kernel", [&] {
    dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
        points.contiguous().data_ptr<scalar_t>(),
        coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
        coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
        coors_z_max, grid_x, grid_y, grid_z, num_points, num_features, NDim);
  });

  AT_CUDA_CHECK(hipGetLastError());
}
