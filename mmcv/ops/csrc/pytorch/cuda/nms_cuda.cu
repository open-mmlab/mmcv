#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved
#include "nms_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

Tensor NMSCUDAKernelLauncher(Tensor boxes, Tensor scores, float iou_threshold,
                             int offset) {
  at::cuda::CUDAGuard device_guard(boxes.device());

  if (boxes.numel() == 0) {
    return at::empty({0}, boxes.options().dtype(at::kLong));
  }
  auto order_t = std::get<1>(scores.sort(0, /*descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);
  const int col_blocks = (boxes_num + threadsPerBlock - 1) / threadsPerBlock;
  const int col_blocks_alloc = GET_BLOCKS(boxes_num, threadsPerBlock);
  Tensor mask =
      at::empty({boxes_num, col_blocks}, boxes.options().dtype(at::kLong));
  dim3 blocks(col_blocks_alloc, col_blocks_alloc);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  nms_cuda<<<blocks, threads, 0, stream>>>(
      boxes_num, iou_threshold, offset, boxes_sorted.data_ptr<float>(),
      (unsigned long long*)mask.data_ptr<int64_t>());

  // Filter the boxes which should be kept.
  at::Tensor keep_t = at::zeros(
      {boxes_num}, boxes.options().dtype(at::kBool).device(at::kCUDA));
  gather_keep_from_mask<<<1, min(col_blocks, THREADS_PER_BLOCK),
                          col_blocks * sizeof(unsigned long long), stream>>>(
      keep_t.data_ptr<bool>(), (unsigned long long*)mask.data_ptr<int64_t>(),
      boxes_num);
  AT_CUDA_CHECK(hipGetLastError());
  return order_t.masked_select(keep_t);
}
