#include <stdio.h>
#include <stdlib.h>

#include "gather_points_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void GatherPointsForwardCUDAKernelLauncher(int b, int c, int n, int npoints,
                                           const Tensor points,
                                           const Tensor idx, Tensor out) {
  // points: (B, C, N)
  // idx: (B, npoints)
  // output:
  //      out: (B, C, npoints)

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c, b);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      points.scalar_type(), "gather_points_forward_cuda_kernel", [&] {
        gather_points_forward_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                b, c, n, npoints, points.data_ptr<scalar_t>(),
                idx.data_ptr<int>(), out.data_ptr<scalar_t>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void GatherPointsBackwardCUDAKernelLauncher(int b, int c, int n, int npoints,
                                            const Tensor grad_out,
                                            const Tensor idx,
                                            Tensor grad_points) {
  // grad_out: (B, C, npoints)
  // idx: (B, npoints)
  // output:
  //      grad_points: (B, C, N)

  at::cuda::CUDAGuard device_guard(grad_out.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c, b);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_out.scalar_type(), "gather_points_backward_cuda_kernel", [&] {
        gather_points_backward_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                b, c, n, npoints, grad_out.data_ptr<scalar_t>(),
                idx.data_ptr<int>(), grad_points.data_ptr<scalar_t>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}
