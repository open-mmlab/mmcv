// Modified from
// https://github.com/sshaoshuai/PCDet/blob/master/pcdet/ops/roiaware_pool3d/src/roiaware_pool3d_kernel.cu
// Written by Shaoshuai Shi
// All Rights Reserved 2019.

#include <stdio.h>

#include "points_in_boxes_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

void PointsInBoxesPartForwardCUDAKernelLauncher(int batch_size, int boxes_num,
                                                int pts_num, const Tensor boxes,
                                                const Tensor pts,
                                                Tensor box_idx_of_points) {
  // params boxes: (B, N, 7) [x, y, z, x_size, y_size, z_size, rz] in LiDAR
  // coordinate, z is
  // the bottom center, each box DO NOT overlaps params pts: (B, npoints, 3) [x,
  // y, z] in LiDAR coordinate params boxes_idx_of_points: (B, npoints), default
  // -1

  at::cuda::CUDAGuard device_guard(boxes.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks(GET_BLOCKS(pts_num, THREADS_PER_BLOCK), batch_size);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      boxes.scalar_type(), "points_in_boxes_part_forward_cuda_kernel", [&] {
        points_in_boxes_part_forward_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                batch_size, boxes_num, pts_num, boxes.data_ptr<scalar_t>(),
                pts.data_ptr<scalar_t>(), box_idx_of_points.data_ptr<int>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void PointsInBoxesAllForwardCUDAKernelLauncher(int batch_size, int boxes_num,
                                               int pts_num, const Tensor boxes,
                                               const Tensor pts,
                                               Tensor box_idx_of_points) {
  // params boxes: (B, N, 7) [x, y, z, x_size, y_size, z_size, rz] in LiDAR
  // coordinate, z is the bottom center, each box params pts: (B, npoints, 3)
  // [x, y, z] in LiDAR coordinate params boxes_idx_of_points: (B, npoints),
  // default -1

  at::cuda::CUDAGuard device_guard(boxes.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 blocks(GET_BLOCKS(pts_num, THREADS_PER_BLOCK), batch_size);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      boxes.scalar_type(), "points_in_boxes_all_forward_cuda_kernel", [&] {
        points_in_boxes_all_forward_cuda_kernel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                batch_size, boxes_num, pts_num, boxes.data_ptr<scalar_t>(),
                pts.data_ptr<scalar_t>(), box_idx_of_points.data_ptr<int>());
      });

  AT_CUDA_CHECK(hipGetLastError());
}
