// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved
#include "pytorch_cuda_helper.hpp"
#include "nms_rotated_cuda.cuh"

Tensor nms_rotated_cuda(
    const Tensor dets,
    const Tensor scores,
    float iou_threshold) {
  // using scalar_t = float;
  AT_ASSERTM(dets.type().is_cuda(), "dets must be a CUDA tensor");
  AT_ASSERTM(scores.type().is_cuda(), "scores must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(dets.device());

  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto dets_sorted = dets.index_select(0, order_t);

  int dets_num = dets.size(0);

  const int col_blocks = at::cuda::ATenCeilDiv(dets_num, threadsPerBlock);

  Tensor mask =
      at::empty({dets_num * col_blocks}, dets.options().dtype(at::kLong));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      dets_sorted.type(), "nms_rotated_kernel_cuda", [&] {
        nms_rotated_cuda_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            dets_num,
            iou_threshold,
            dets_sorted.data<scalar_t>(),
            (unsigned long long*)mask.data<int64_t>());
      });

  Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host = (unsigned long long*)mask_cpu.data<int64_t>();

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  Tensor keep =
      at::empty({dets_num}, dets.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < dets_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  AT_CUDA_CHECK(hipGetLastError());
  return order_t.index(
      {keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep)
           .to(order_t.device(), keep.scalar_type())});
}
