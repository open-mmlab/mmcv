#include "pytorch_cuda_helper.hpp"

#include "carafe_cuda_kernel.cuh"

void CARAFEForwardCUDAKernelLauncher(const Tensor features, const Tensor masks,
                                     Tensor rfeatures, Tensor routput,
                                     Tensor rmasks, Tensor output,
                                     const int kernel_size,
                                     const int group_size,
                                     const int scale_factor) {
  const int batch_size = output.size(0);
  const int channels = output.size(1);
  const int output_height = output.size(2);
  const int output_width = output.size(3);

  const int input_height = features.size(2);
  const int input_width = features.size(3);

  const int mask_channels = masks.size(1);

  rfeatures.resize_({batch_size, input_height, input_width, channels});
  routput.resize_({batch_size, output_height, output_width, channels});
  rmasks.resize_({batch_size, output_height, output_width, mask_channels});

  // one warp per pixel
  at::cuda::CUDAGuard device_guard(features.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "NCHW2NHWC_Feature", ([&] {
        const scalar_t *bottom_data = features.data_ptr<scalar_t>();
        scalar_t *top_data = rfeatures.data_ptr<scalar_t>();
        const int dh = divideUP(channels, kTileDim);
        const int dw = divideUP(input_height * input_width, kTileDim);
        BatchTranspose2DCUDAKernel<scalar_t>
            <<<batch_size * dh * dw, dim3(kTileDim, kBlockRows), 0, stream>>>(
                batch_size, channels, input_height * input_width, dh, dw,
                bottom_data, top_data);
      }));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "NCHW2NHWC_Masks", ([&] {
        const scalar_t *bottom_data = masks.data_ptr<scalar_t>();
        scalar_t *top_data = rmasks.data_ptr<scalar_t>();
        const int dh = divideUP(mask_channels, kTileDim);
        const int dw = divideUP(output_height * output_width, kTileDim);
        BatchTranspose2DCUDAKernel<scalar_t>
            <<<batch_size * dh * dw, dim3(kTileDim, kBlockRows), 0, stream>>>(
                batch_size, mask_channels, output_height * output_width, dh, dw,
                bottom_data, top_data);
      }));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "CARAFELaucherForward", ([&] {
        const int num_kernels =
            batch_size * output_height * output_width * THREADS_PER_PIXEL;
        const scalar_t *bottom_data = rfeatures.data_ptr<scalar_t>();
        const scalar_t *bottom_masks = rmasks.data_ptr<scalar_t>();
        scalar_t *top_data = routput.data_ptr<scalar_t>();

        CARAFEForward<scalar_t><<<divideUP(num_kernels, THREADS_PER_BLOCK),
                                  THREADS_PER_BLOCK, 0, stream>>>(
            num_kernels, bottom_data, bottom_masks, kernel_size, group_size,
            scale_factor, channels, input_height, input_width, output_height,
            output_width, mask_channels, top_data);
      }));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.scalar_type(), "NHWC2NCHW", ([&] {
        const scalar_t *bottom_data = routput.data_ptr<scalar_t>();
        scalar_t *top_data = output.data_ptr<scalar_t>();
        const int dh = divideUP(output_height * output_width, kTileDim);
        const int dw = divideUP(channels, kTileDim);
        BatchTranspose2DCUDAKernel<scalar_t>
            <<<batch_size * dh * dw, dim3(kTileDim, kBlockRows), 0, stream>>>(
                batch_size, output_height * output_width, channels, dh, dw,
                bottom_data, top_data);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}

void CARAFEBackwardCUDAKernelLauncher(
    const Tensor top_grad, const Tensor rfeatures, const Tensor masks,
    Tensor rtop_grad, Tensor rbottom_grad_hs, Tensor rbottom_grad,
    Tensor rmask_grad, Tensor bottom_grad, Tensor mask_grad,
    const int kernel_size, const int group_size, const int scale_factor) {
  const int batch_size = top_grad.size(0);
  const int channels = top_grad.size(1);
  const int output_height = top_grad.size(2);
  const int output_width = top_grad.size(3);

  const int input_height = bottom_grad.size(2);
  const int input_width = bottom_grad.size(3);

  const int mask_channels = masks.size(1);

  rtop_grad.resize_({batch_size, output_height, output_width, channels});
  rbottom_grad.resize_({batch_size, input_height, input_width, channels});
  rbottom_grad_hs.resize_({batch_size, output_height, output_width, channels});
  rmask_grad.resize_({batch_size, output_height, output_width, mask_channels});

  at::cuda::CUDAGuard device_guard(top_grad.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "NCHW2NHWC_Top_Grad", ([&] {
        const scalar_t *bottom_data = top_grad.data_ptr<scalar_t>();
        scalar_t *top_data = rtop_grad.data_ptr<scalar_t>();
        const int dh = divideUP(channels, kTileDim);
        const int dw = divideUP(output_height * output_width, kTileDim);
        BatchTranspose2DCUDAKernel<scalar_t>
            <<<batch_size * dh * dw, dim3(kTileDim, kBlockRows), 0, stream>>>(
                batch_size, channels, output_height * output_width, dh, dw,
                bottom_data, top_data);
      }));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "CARAFELaucherBackward_Feature", ([&] {
        const int num_kernels =
            batch_size * output_height * output_width * THREADS_PER_PIXEL;
        const scalar_t *top_diff = rtop_grad.data_ptr<scalar_t>();
        const scalar_t *bottom_masks = masks.data_ptr<scalar_t>();
        scalar_t *bottom_diff = rbottom_grad_hs.data_ptr<scalar_t>();

        CARAFEBackward_Feature<scalar_t>
            <<<divideUP(num_kernels, THREADS_PER_BLOCK), THREADS_PER_BLOCK, 0,
               stream>>>(num_kernels, top_diff, bottom_masks, kernel_size,
                         group_size, scale_factor, channels, input_height,
                         input_width, output_height, output_width,
                         mask_channels, bottom_diff);
      }));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "FeatureSum", ([&] {
        const int num_kernels =
            batch_size * input_height * input_width * THREADS_PER_PIXEL;
        const scalar_t *bottom_diff_hs = rbottom_grad_hs.data_ptr<scalar_t>();
        scalar_t *bottom_diff = rbottom_grad.data_ptr<scalar_t>();

        FeatureSum<scalar_t>
            <<<divideUP(num_kernels, THREADS_PER_BLOCK), THREADS_PER_BLOCK, 0,
               stream>>>(num_kernels, bottom_diff_hs, scale_factor, channels,
                         input_height, input_width, bottom_diff);
      }));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "NHWC2NCHW_Bottom_Grad", ([&] {
        const scalar_t *bottom_data = rbottom_grad.data_ptr<scalar_t>();
        scalar_t *top_data = bottom_grad.data_ptr<scalar_t>();
        const int dh = divideUP(input_height * input_width, kTileDim);
        const int dw = divideUP(channels, kTileDim);
        BatchTranspose2DCUDAKernel<scalar_t>
            <<<batch_size * dh * dw, dim3(kTileDim, kBlockRows), 0, stream>>>(
                batch_size, input_height * input_width, channels, dh, dw,
                bottom_data, top_data);
      }));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "CARAFELaucherBackward_Mask", ([&] {
        const int num_kernels = batch_size * output_height * output_width *
                                mask_channels * WARP_SIZE;
        const scalar_t *top_diff = rtop_grad.data_ptr<scalar_t>();
        const scalar_t *bottom_data = rfeatures.data_ptr<scalar_t>();
        scalar_t *mask_diff = rmask_grad.data_ptr<scalar_t>();

        CARAFEBackward_Mask<scalar_t>
            <<<divideUP(num_kernels, THREADS_PER_BLOCK), THREADS_PER_BLOCK, 0,
               stream>>>(num_kernels, top_diff, bottom_data, kernel_size,
                         group_size, scale_factor, channels, input_height,
                         input_width, output_height, output_width,
                         mask_channels, mask_diff);
      }));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.scalar_type(), "NHWC2NCHW_Mask_Grad", ([&] {
        const scalar_t *bottom_data = rmask_grad.data_ptr<scalar_t>();
        scalar_t *top_data = mask_grad.data_ptr<scalar_t>();
        const int dh = divideUP(output_height * output_width, kTileDim);
        const int dw = divideUP(mask_channels, kTileDim);
        BatchTranspose2DCUDAKernel<scalar_t>
            <<<batch_size * dh * dw, dim3(kTileDim, kBlockRows), 0, stream>>>(
                batch_size, output_height * output_width, mask_channels, dh, dw,
                bottom_data, top_data);
      }));

  AT_CUDA_CHECK(hipGetLastError());
}
